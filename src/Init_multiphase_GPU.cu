﻿#include "externLib.h"
#include "solver_precision.h"
#include "externLib_CUDA.cuh"
#include "Module_extern.h"
#include "utils_GPU.cuh"
#include "Init_multiphase_GPU.h"
#include "Fluid_singlephase_extern.h"
#include "Fluid_multiphase_extern.h"
#include "Idx_cpu.h"


/* initialization basic - CUDA */
void initialization_GPU() {
    cout << "***************************** GPU Specifications **********************************" << endl;
    const int kb = 1024;
    const int mb = kb * kb;
    hipDeviceProp_t props;
    hipGetDeviceProperties(&props, 0);
    cout << 0 << ": " << props.name << ": " << props.major << "." << props.minor << endl;
    cout << "  Global memory:   " << props.totalGlobalMem / mb << " mb" << endl;
    cout << "  Shared memory:   " << props.sharedMemPerBlock / kb << " kb" << endl;
    cout << "  Constant memory: " << props.totalConstMem / kb << " kb" << endl;
    cout << "  Block registers: " << props.regsPerBlock << endl;
    cout << "  Number of SMs: " << props.multiProcessorCount << endl;
    cout << "  Clock frequencey: " << props.clockRate / 1e3 << " MHz" << endl;
    cout << "  Warp size:         " << props.warpSize << endl;
    cout << "  Threads per block: " << props.maxThreadsPerBlock << endl;
    cout << "  Max block dimensions: [ " << props.maxThreadsDim[0] << ", " << props.maxThreadsDim[1] << ", " << props.maxThreadsDim[2] << " ]" << endl;
    cout << "  Max grid dimensions:  [ " << props.maxGridSize[0] << ", " << props.maxGridSize[1] << ", " << props.maxGridSize[2] << " ]" << endl;
    cout << "***************************** GPU Specifications **********************************" << endl;

    // allocate memory for (walls) and copy data from host
    MemAllocate_geometry_GPU(1);
    //************* fluid flow related memory allocate/deallocate ******************************
    MemAllocate_multi_GPU(1);

    cout << "Total amount of global memory needed on GPU (GB) = " << (memory_gpu / double(1024 * 1024 * 1024)) << endl;

}


/* geometry related memory allocate/deallocate */
void MemAllocate_geometry_GPU(int flag) {
    if (flag == 1) {
        /* allocate memory on device */
        cudaErrorCheck(hipMalloc(&walls_d, mem_size_s2_int)); memory_gpu += mem_size_s2_int;
        cudaErrorCheck(hipMemcpy(walls_d, walls, mem_size_s2_int, hipMemcpyHostToDevice));

        cudaErrorCheck(hipMalloc(&walls_type_d, mem_size_s4_int)); memory_gpu += mem_size_s4_int;
        cudaErrorCheck(hipMemcpy(walls_type_d, walls_type, mem_size_s4_int, hipMemcpyHostToDevice));

        cudaErrorCheck(hipMalloc(&s_nx_d, mem_size_s4_TP)); memory_gpu += mem_size_s4_TP;
        cudaErrorCheck(hipMemcpy(s_nx_d, s_nx, mem_size_s4_TP, hipMemcpyHostToDevice));

        cudaErrorCheck(hipMalloc(&s_ny_d, mem_size_s4_TP)); memory_gpu += mem_size_s4_TP;
        cudaErrorCheck(hipMemcpy(s_ny_d, s_ny, mem_size_s4_TP, hipMemcpyHostToDevice));

        cudaErrorCheck(hipMalloc(&s_nz_d, mem_size_s4_TP)); memory_gpu += mem_size_s4_TP;
        cudaErrorCheck(hipMemcpy(s_nz_d, s_nz, mem_size_s4_TP, hipMemcpyHostToDevice));
        
    }
    else {
        cudaErrorCheck(hipFree(walls_d));
        cudaErrorCheck(hipFree(walls_type_d));
        cudaErrorCheck(hipFree(s_nx_d));
        cudaErrorCheck(hipFree(s_ny_d));
        cudaErrorCheck(hipFree(s_nz_d));
    }

}

// ************* fluid flow related memory allocate/deallocate ******************************
void MemAllocate_multi_GPU(int flag) {
    if (flag == 1) {
        cudaErrorCheck(hipMalloc(&u_d, mem_size_s1_TP)); memory_gpu += mem_size_s1_TP; cudaErrorCheck(hipMemcpy(u_d, u, mem_size_s1_TP, hipMemcpyHostToDevice));
        cudaErrorCheck(hipMalloc(&v_d, mem_size_s1_TP)); memory_gpu += mem_size_s1_TP; cudaErrorCheck(hipMemcpy(v_d, v, mem_size_s1_TP, hipMemcpyHostToDevice));
        cudaErrorCheck(hipMalloc(&w_d, mem_size_s1_TP)); memory_gpu += mem_size_s1_TP; cudaErrorCheck(hipMemcpy(w_d, w, mem_size_s1_TP, hipMemcpyHostToDevice));
        cudaErrorCheck(hipMalloc(&rho_d, mem_size_s1_TP)); memory_gpu += mem_size_s1_TP; cudaErrorCheck(hipMemcpy(rho_d, rho, mem_size_s1_TP, hipMemcpyHostToDevice));
        cudaErrorCheck(hipMalloc(&curv_d, mem_size_s1_TP)); memory_gpu += mem_size_s1_TP; cudaErrorCheck(hipMemcpy(curv_d, curv, mem_size_s1_TP, hipMemcpyHostToDevice));
        cudaErrorCheck(hipMalloc(&W_in_d, NXG1 * NYG1 * sizeof(T_P))); memory_gpu += NXG1 * NYG1 * sizeof(T_P); cudaErrorCheck(hipMemcpy(W_in_d, W_in, NXG1 * NYG1 * sizeof(T_P), hipMemcpyHostToDevice));
        cudaErrorCheck(hipMalloc(&pdf_d, mem_size_f1_TP)); memory_gpu += mem_size_f1_TP; cudaErrorCheck(hipMemcpy(pdf_d, pdf, mem_size_f1_TP, hipMemcpyHostToDevice));
        cudaErrorCheck(hipMalloc(&cn_x_d, mem_size_s2_TP)); memory_gpu += mem_size_s2_TP; cudaErrorCheck(hipMemcpy(cn_x_d, cn_x, mem_size_s2_TP, hipMemcpyHostToDevice));
        cudaErrorCheck(hipMalloc(&cn_y_d, mem_size_s2_TP)); memory_gpu += mem_size_s2_TP; cudaErrorCheck(hipMemcpy(cn_y_d, cn_y, mem_size_s2_TP, hipMemcpyHostToDevice));
        cudaErrorCheck(hipMalloc(&cn_z_d, mem_size_s2_TP)); memory_gpu += mem_size_s2_TP; cudaErrorCheck(hipMemcpy(cn_z_d, cn_z, mem_size_s2_TP, hipMemcpyHostToDevice));
        cudaErrorCheck(hipMalloc(&c_norm_d, mem_size_s2_TP)); memory_gpu += mem_size_s2_TP; cudaErrorCheck(hipMemcpy(c_norm_d, c_norm, mem_size_s2_TP, hipMemcpyHostToDevice));
        //convective BC
        if (outlet_BC == 1) {
            cudaErrorCheck(hipMalloc(&phi_convec_bc_d, NXG1 * NYG1 * sizeof(T_P))); memory_gpu += NXG1 * NYG1 * sizeof(T_P); cudaErrorCheck(hipMemcpy(phi_convec_bc_d, phi_convec_bc, NXG1 * NYG1 * sizeof(T_P), hipMemcpyHostToDevice));
            cudaErrorCheck(hipMalloc(&g_convec_bc_d, NXG1 * NYG1 * 19 * sizeof(T_P))); memory_gpu += NXG1 * NYG1 * 19 * sizeof(T_P); cudaErrorCheck(hipMemcpy(g_convec_bc_d, g_convec_bc, NXG1 * NYG1 * 19 * sizeof(T_P), hipMemcpyHostToDevice));
            cudaErrorCheck(hipMalloc(&f_convec_bc_d, NXG1 * NYG1 * 19 * sizeof(T_P))); memory_gpu += NXG1 * NYG1 * 19 * sizeof(T_P); cudaErrorCheck(hipMemcpy(f_convec_bc_d, f_convec_bc, NXG1 * NYG1 * 19 * sizeof(T_P), hipMemcpyHostToDevice));
        }
        cudaErrorCheck(hipMalloc(&phi_d, mem_size_s4_TP)); memory_gpu += mem_size_s4_TP; cudaErrorCheck(hipMemcpy(phi_d, phi, mem_size_s4_TP, hipMemcpyHostToDevice));

    }
    else {
        cudaErrorCheck(hipFree(u_d));
        cudaErrorCheck(hipFree(v_d));
        cudaErrorCheck(hipFree(w_d));
        //cudaErrorCheck(hipFree(rho_d));
        cudaErrorCheck(hipFree(curv_d));
        cudaErrorCheck(hipFree(W_in_d));
        cudaErrorCheck(hipFree(pdf_d));
        cudaErrorCheck(hipFree(cn_x_d));
        cudaErrorCheck(hipFree(cn_y_d));
        cudaErrorCheck(hipFree(cn_z_d));
        cudaErrorCheck(hipFree(c_norm_d));
        if (outlet_BC == 1) {
            cudaErrorCheck(hipFree(phi_convec_bc_d));
            cudaErrorCheck(hipFree(g_convec_bc_d));
            cudaErrorCheck(hipFree(f_convec_bc_d));
        }
        cudaErrorCheck(hipFree(phi_d));


    }
}
