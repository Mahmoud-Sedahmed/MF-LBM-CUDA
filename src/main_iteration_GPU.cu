#include "hip/hip_runtime.h"
﻿#include "externLib.h"
#include "solver_precision.h"
#include "externLib_CUDA.cuh"
#include "main_iteration_GPU.h"
#include "Module_extern.h"
#include "utils.h"
#include "utils_GPU.cuh"
#include "Fluid_singlephase_extern.h"
#include "Fluid_multiphase_extern.h"
#include "Global_Variables_GPU.cuh"
#include "Idx_gpu.cuh"

/* copy constant data to GPU */
void copyConstantData() {
    cudaErrorCheck(hipMemcpyToSymbol(HIP_SYMBOL(lbm_gamma_d), &lbm_gamma, sizeof(T_P)));
    cudaErrorCheck(hipMemcpyToSymbol(HIP_SYMBOL(force_z_d), &force_z, sizeof(T_P)));
    cudaErrorCheck(hipMemcpyToSymbol(HIP_SYMBOL(la_nui1_d), &la_nui1, sizeof(T_P)));
    cudaErrorCheck(hipMemcpyToSymbol(HIP_SYMBOL(la_nui2_d), &la_nui2, sizeof(T_P)));
    cudaErrorCheck(hipMemcpyToSymbol(HIP_SYMBOL(mrt_e2_coef1_d), &mrt_e2_coef1, sizeof(T_P)));
    cudaErrorCheck(hipMemcpyToSymbol(HIP_SYMBOL(mrt_e2_coef2_d), &mrt_e2_coef2, sizeof(T_P)));
    cudaErrorCheck(hipMemcpyToSymbol(HIP_SYMBOL(mrt_omega_xx_d), &mrt_omega_xx, sizeof(T_P)));
    cudaErrorCheck(hipMemcpyToSymbol(HIP_SYMBOL(mrt_coef1_d), &mrt_coef1, sizeof(T_P)));
    cudaErrorCheck(hipMemcpyToSymbol(HIP_SYMBOL(mrt_coef2_d), &mrt_coef2, sizeof(T_P)));
    cudaErrorCheck(hipMemcpyToSymbol(HIP_SYMBOL(mrt_coef3_d), &mrt_coef3, sizeof(T_P)));
    cudaErrorCheck(hipMemcpyToSymbol(HIP_SYMBOL(mrt_coef4_d), &mrt_coef4, sizeof(T_P)));
    cudaErrorCheck(hipMemcpyToSymbol(HIP_SYMBOL(lbm_beta_d), &lbm_beta, sizeof(T_P)));
    cudaErrorCheck(hipMemcpyToSymbol(HIP_SYMBOL(w_equ_1_d), &w_equ_1, sizeof(T_P)));
    cudaErrorCheck(hipMemcpyToSymbol(HIP_SYMBOL(w_equ_2_d), &w_equ_2, sizeof(T_P)));
    cudaErrorCheck(hipMemcpyToSymbol(HIP_SYMBOL(RK_weight2_d), &RK_weight2, sizeof(T_P)));
    cudaErrorCheck(hipMemcpyToSymbol(HIP_SYMBOL(w_equ_d), w_equ, 19 * sizeof(T_P)));
    cudaErrorCheck(hipMemcpyToSymbol(HIP_SYMBOL(ex_d), ex, 19 * sizeof(int)));
    cudaErrorCheck(hipMemcpyToSymbol(HIP_SYMBOL(ey_d), ey, 19 * sizeof(int)));
    cudaErrorCheck(hipMemcpyToSymbol(HIP_SYMBOL(ez_d), ez, 19 * sizeof(int)));
    cudaErrorCheck(hipMemcpyToSymbol(HIP_SYMBOL(Z_porous_plate_d), &Z_porous_plate, sizeof(int)));
    cudaErrorCheck(hipMemcpyToSymbol(HIP_SYMBOL(porous_plate_cmd_d), &porous_plate_cmd, sizeof(int)));
    cudaErrorCheck(hipMemcpyToSymbol(HIP_SYMBOL(num_solid_boundary_d), &num_solid_boundary, sizeof(long long)));
    cudaErrorCheck(hipMemcpyToSymbol(HIP_SYMBOL(num_fluid_boundary_d), &num_fluid_boundary, sizeof(long long)));
    cudaErrorCheck(hipMemcpyToSymbol(HIP_SYMBOL(ISO4_d), ISO4, 2 * sizeof(T_P)));
    cudaErrorCheck(hipMemcpyToSymbol(HIP_SYMBOL(nxGlobal_d), &nxGlobal, sizeof(long long)));
    cudaErrorCheck(hipMemcpyToSymbol(HIP_SYMBOL(nyGlobal_d), &nyGlobal, sizeof(long long)));
    cudaErrorCheck(hipMemcpyToSymbol(HIP_SYMBOL(nzGlobal_d), &nzGlobal, sizeof(long long)));
    cudaErrorCheck(hipMemcpyToSymbol(HIP_SYMBOL(phi_inlet_d), &phi_inlet, sizeof(T_P)));
    cudaErrorCheck(hipMemcpyToSymbol(HIP_SYMBOL(relaxation_d), &relaxation, sizeof(T_P)));
    cudaErrorCheck(hipMemcpyToSymbol(HIP_SYMBOL(sa_inject_d), &sa_inject, sizeof(T_P)));
    cudaErrorCheck(hipMemcpyToSymbol(HIP_SYMBOL(uin_avg_d), &uin_avg, sizeof(T_P)));
    cudaErrorCheck(hipMemcpyToSymbol(HIP_SYMBOL(cos_theta_d), &cos_theta, sizeof(T_P)));
}



#pragma region (kernel_multiphase)
//=====================================================================================================================================
//----------------------odd step kernel----------------------
// complete two streaming steps
//=====================================================================================================================================
__global__ void kernel_odd_color_GPU(int ixmin, int ixmax, int iymin, int iymax, int izmin, int izmax,
    int* walls, T_P* pdf, T_P* phi, T_P* cn_x, T_P* cn_y, T_P* cn_z, T_P* curv, T_P* c_norm) {

    // Indexing (Thread)
    int i = threadIdx.x + blockIdx.x * blockDim.x + 1;
    int j = threadIdx.y + blockIdx.y * blockDim.y + 1;
    int k = threadIdx.z + blockIdx.z * blockDim.z + 1;

    T_P fx, fy, fz, omega, cnx, cny, cnz;
    T_P sum1, sum2, sum3, sum4, sum5, sum6, sum7, sum8, sum9;
    T_P m_rho, m_e, m_e2, m_jx, m_qx, m_jy, m_qy, m_jz, m_qz, m_3pxx, m_3pixx, m_pww, m_piww, m_pxy, m_pyz, m_pzx, m_tx, m_ty, m_tz;
    T_P ft0, ft1, ft2, ft3, ft4, ft5, ft6, ft7, ft8, ft9, ft10, ft11, ft12, ft13, ft14, ft15, ft16, ft17, ft18;
    T_P g1t0, g1t1, g1t2, g1t3, g1t4, g1t5, g1t6, g1t7, g1t8, g1t9, g1t10, g1t11, g1t12, g1t13, g1t14, g1t15, g1t16, g1t17, g1t18;
    T_P g2t0, g2t1, g2t2, g2t3, g2t4, g2t5, g2t6, g2t7, g2t8, g2t9, g2t10, g2t11, g2t12, g2t13, g2t14, g2t15, g2t16, g2t17, g2t18;
    T_P s_e, s_e2, s_q, s_nu, s_pi, s_t;  //relaxation parameters
    T_P ux1, uy1, uz1, den, u2, rho1, rho2, tmp, tmp1;

    if (walls[i_s2(i, j, k)] == 0 && i >= ixmin && i <= ixmax && j >= iymin && j <= iymax && k >= izmin && k <= izmax) {
        //++++++++ + -AA pattern pull step++++++++++++
        g1t0 = pdf[i_f1(i, j, k, 0, 0)];
        g1t1 = pdf[i_f1(i - 1, j, k, 1, 0)];
        g1t2 = pdf[i_f1(i + 1, j, k, 2, 0)];
        g1t3 = pdf[i_f1(i, j - 1, k, 3, 0)];
        g1t4 = pdf[i_f1(i, j + 1, k, 4, 0)];
        g1t5 = pdf[i_f1(i, j, k - 1, 5, 0)];
        g1t6 = pdf[i_f1(i, j, k + 1, 6, 0)];
        g1t7 = pdf[i_f1(i - 1, j - 1, k, 7, 0)];
        g1t8 = pdf[i_f1(i + 1, j - 1, k, 8, 0)];
        g1t9 = pdf[i_f1(i - 1, j + 1, k, 9, 0)];
        g1t10 = pdf[i_f1(i + 1, j + 1, k, 10, 0)];
        g1t11 = pdf[i_f1(i - 1, j, k - 1, 11, 0)];
        g1t12 = pdf[i_f1(i + 1, j, k - 1, 12, 0)];
        g1t13 = pdf[i_f1(i - 1, j, k + 1, 13, 0)];
        g1t14 = pdf[i_f1(i + 1, j, k + 1, 14, 0)];
        g1t15 = pdf[i_f1(i, j - 1, k - 1, 15, 0)];
        g1t16 = pdf[i_f1(i, j + 1, k - 1, 16, 0)];
        g1t17 = pdf[i_f1(i, j - 1, k + 1, 17, 0)];
        g1t18 = pdf[i_f1(i, j + 1, k + 1, 18, 0)];

        g2t0 = pdf[i_f1(i, j, k, 0, 1)];
        g2t1 = pdf[i_f1(i - 1, j, k, 1, 1)];
        g2t2 = pdf[i_f1(i + 1, j, k, 2, 1)];
        g2t3 = pdf[i_f1(i, j - 1, k, 3, 1)];
        g2t4 = pdf[i_f1(i, j + 1, k, 4, 1)];
        g2t5 = pdf[i_f1(i, j, k - 1, 5, 1)];
        g2t6 = pdf[i_f1(i, j, k + 1, 6, 1)];
        g2t7 = pdf[i_f1(i - 1, j - 1, k, 7, 1)];
        g2t8 = pdf[i_f1(i + 1, j - 1, k, 8, 1)];
        g2t9 = pdf[i_f1(i - 1, j + 1, k, 9, 1)];
        g2t10 = pdf[i_f1(i + 1, j + 1, k, 10, 1)];
        g2t11 = pdf[i_f1(i - 1, j, k - 1, 11, 1)];
        g2t12 = pdf[i_f1(i + 1, j, k - 1, 12, 1)];
        g2t13 = pdf[i_f1(i - 1, j, k + 1, 13, 1)];
        g2t14 = pdf[i_f1(i + 1, j, k + 1, 14, 1)];
        g2t15 = pdf[i_f1(i, j - 1, k - 1, 15, 1)];
        g2t16 = pdf[i_f1(i, j + 1, k - 1, 16, 1)];
        g2t17 = pdf[i_f1(i, j - 1, k + 1, 17, 1)];
        g2t18 = pdf[i_f1(i, j + 1, k + 1, 18, 1)];

        // let ft be the bulk PDF
        ft0 = g1t0 + g2t0;
        ft1 = g1t1 + g2t1;
        ft2 = g1t2 + g2t2;
        ft3 = g1t3 + g2t3;
        ft4 = g1t4 + g2t4;
        ft5 = g1t5 + g2t5;
        ft6 = g1t6 + g2t6;
        ft7 = g1t7 + g2t7;
        ft8 = g1t8 + g2t8;
        ft9 = g1t9 + g2t9;
        ft10 = g1t10 + g2t10;
        ft11 = g1t11 + g2t11;
        ft12 = g1t12 + g2t12;
        ft13 = g1t13 + g2t13;
        ft14 = g1t14 + g2t14;
        ft15 = g1t15 + g2t15;
        ft16 = g1t16 + g2t16;
        ft17 = g1t17 + g2t17;
        ft18 = g1t18 + g2t18;

        // order parameter
        rho1 = g1t0 + g1t1 + g1t2 + g1t3 + g1t4 + g1t5 + g1t6 + g1t7 + g1t8 + g1t9 + g1t10 + g1t11 + g1t12 + g1t13 + g1t14 + g1t15 + g1t16 + g1t17 + g1t18;
        rho2 = g2t0 + g2t1 + g2t2 + g2t3 + g2t4 + g2t5 + g2t6 + g2t7 + g2t8 + g2t9 + g2t10 + g2t11 + g2t12 + g2t13 + g2t14 + g2t15 + g2t16 + g2t17 + g2t18;

        T_P phi_loc = (rho1 - rho2) / (rho1 + rho2);
        phi[i_s4(i, j, k)] = phi_loc;

        cnx = cn_x[i_s2(i, j, k)];
        cny = cn_y[i_s2(i, j, k)];
        cnz = cn_z[i_s2(i, j, k)];

        tmp = prc(0.5) * lbm_gamma_d * curv[i_s1(i, j, k)] * c_norm[i_s2(i, j, k)];
        fx = tmp * cnx;
        fy = tmp * cny;
        fz = tmp * cnz + force_z_d;   //body force force_z along flow direction

        //++++++++++++ - MRT COLLISION++++++++++++ + -
        // select viscosity++++++++++++++++++++++ + -
        omega = prc(1.) / (prc(6.) / ((prc(1.0) + phi_loc) * la_nui1_d + (prc(1.0) - phi_loc) * la_nui2_d) + prc(0.5));
        //MRT PARAMETERS
        s_nu = omega;
#if (mrt==1)
        //************bounceback opt * ***********
        s_e = omega;
        s_e2 = omega;
        s_pi = omega;
        s_q = prc(8.) * (prc(2.) - omega) / (prc(8.) - omega);
        s_t = s_q;
#elif (mrt==2)
        //************original * ***********
        s_e = prc(1.19);
        s_e2 = prc(1.4);
        s_pi = prc(1.4);
        s_q = prc(1.2);
        s_t = prc(1.98);

#elif (mrt==3 )           
        //************SRT * ***********
        s_e = omega;
        s_e2 = omega;
        s_pi = omega;
        s_q = omega;
        s_t = omega;
#elif (mrt==4 )           
        //************advection opt * ***********
        s_e = omega;
        s_e2 = omega;
        s_pi = omega;
        s_q = (prc(6.) - prc(3.) * omega) / (prc(3.) - omega);
        s_t = omega;
#endif 
        //~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~MRT kernel, repeated part~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~
        //++++++++ + -!calculate macroscopic variables++++++++++++
        den = rho1 + rho2;
        ux1 = ft1 - ft2 + ft7 - ft8 + ft9 - ft10 + ft11 - ft12 + ft13 - ft14 + prc(0.5) * fx;
        uy1 = ft3 - ft4 + ft7 + ft8 - ft9 - ft10 + ft15 - ft16 + ft17 - ft18 + prc(0.5) * fy;
        uz1 = ft5 - ft6 + ft11 + ft12 - ft13 - ft14 + ft15 + ft16 - ft17 - ft18 + prc(0.5) * fz;

        u2 = ux1 * ux1 + uy1 * uy1 + uz1 * uz1;

        //PDFs summations for computation efficiency purpose
        sum1 = ft1 + ft2 + ft3 + ft4 + ft5 + ft6;
        sum2 = ft7 + ft8 + ft9 + ft10 + ft11 + ft12 + ft13 + ft14 + ft15 + ft16 + ft17 + ft18;
        sum3 = ft7 - ft8 + ft9 - ft10 + ft11 - ft12 + ft13 - ft14;
        sum4 = ft7 + ft8 - ft9 - ft10 + ft15 - ft16 + ft17 - ft18;
        sum5 = ft11 + ft12 - ft13 - ft14 + ft15 + ft16 - ft17 - ft18;
        sum6 = prc(2.) * (ft1 + ft2) - ft3 - ft4 - ft5 - ft6;
        sum7 = ft7 + ft8 + ft9 + ft10 + ft11 + ft12 + ft13 + ft14 - prc(2.) * (ft15 + ft16 + ft17 + ft18);
        sum8 = ft3 + ft4 - ft5 - ft6;
        sum9 = ft7 + ft8 + ft9 + ft10 - ft11 - ft12 - ft13 - ft14;
        //PDF to moment
        m_rho = den;
        m_e = prc(-30.) * ft0 - prc(11.) * sum1 + prc(8.) * sum2;
        m_e2 = prc(12.) * ft0 - prc(4.) * sum1 + sum2;
        m_jx = ft1 - ft2 + sum3;
        m_qx = prc(-4.) * (ft1 - ft2) + sum3;
        m_jy = ft3 - ft4 + sum4;
        m_qy = prc(-4.) * (ft3 - ft4) + sum4;
        m_jz = ft5 - ft6 + sum5;
        m_qz = prc(-4.) * (ft5 - ft6) + sum5;
        m_3pxx = sum6 + sum7;
        m_3pixx = prc(-2.) * sum6 + sum7;
        m_pww = sum8 + sum9;
        m_piww = prc(-2.) * sum8 + sum9;
        m_pxy = ft7 - ft8 - ft9 + ft10;
        m_pyz = ft15 - ft16 - ft17 + ft18;
        m_pzx = ft11 - ft12 - ft13 + ft14;
        m_tx = ft7 - ft8 + ft9 - ft10 - ft11 + ft12 - ft13 + ft14;
        m_ty = -ft7 - ft8 + ft9 + ft10 + ft15 - ft16 + ft17 - ft18;
        m_tz = ft11 + ft12 - ft13 - ft14 - ft15 - ft16 + ft17 + ft18;

        // relaxtion in moment space
        m_e = m_e - s_e * (m_e - (prc(-11.0) * den + prc(19.0) * u2)) + (prc(38.) - prc(19.) * s_e) * (fx * ux1 + fy * uy1 + fz * uz1);                           //m1
        m_e2 = m_e2 - s_e2 * (m_e2 - (mrt_e2_coef1_d * den + mrt_e2_coef2_d * u2)) + (prc(-11.) + prc(5.5) * s_e2) * (fx * ux1 + fy * uy1 + fz * uz1);                     //m2
        m_jx = m_jx + fx;                                                                                                   //m3
        m_qx = m_qx - s_q * (m_qx - (prc(-0.666666666666666667) * ux1)) + (prc(-0.666666666666666667) + prc(0.333333333333333333) * s_q) * fx; //m4
        m_jy = m_jy + fy;                                                                                                   //m5
        m_qy = m_qy - s_q * (m_qy - (prc(-0.666666666666666667) * uy1)) + (prc(-0.666666666666666667) + prc(0.333333333333333333) * s_q) * fy; //m6
        m_jz = m_jz + fz;                                                                                                   //m7
        m_qz = m_qz - s_q * (m_qz - (prc(-0.666666666666666667) * uz1)) + (prc(-0.666666666666666667) + prc(0.333333333333333333) * s_q) * fz; //m8

        m_3pxx = m_3pxx - s_nu * (m_3pxx - (prc(3.) * ux1 * ux1 - u2)) + (prc(2.) - s_nu) * (prc(2.) * fx * ux1 - fy * uy1 - fz * uz1);                           //m9
        m_3pixx = m_3pixx - s_pi * (m_3pixx - mrt_omega_xx_d * (prc(3.) * ux1 * ux1 - u2)) + (prc(1.) - prc(0.5) * s_pi) * (prc(-2.) * fx * ux1 + fy * uy1 + fz * uz1);         //m10
        m_pww = m_pww - s_nu * (m_pww - (uy1 * uy1 - uz1 * uz1)) + (prc(2.) - s_nu) * (fy * uy1 - fz * uz1);                                     //m11
        m_piww = m_piww - s_pi * (m_piww - mrt_omega_xx_d * (uy1 * uy1 - uz1 * uz1)) + (prc(1.) - prc(0.5) * s_pi) * (-fy * uy1 + fz * uz1);                   //m12
        m_pxy = m_pxy - s_nu * (m_pxy - (ux1 * uy1)) + (prc(1.) - prc(0.5) * s_nu) * (fx * uy1 + fy * ux1);                                        //m13
        m_pyz = m_pyz - s_nu * (m_pyz - (uy1 * uz1)) + (prc(1.) - prc(0.5) * s_nu) * (fy * uz1 + fz * uy1);                                        //m14
        m_pzx = m_pzx - s_nu * (m_pzx - (ux1 * uz1)) + (prc(1.) - prc(0.5) * s_nu) * (fx * uz1 + fz * ux1);                                        //m15
        m_tx = m_tx - s_t * (m_tx); //m16
        m_ty = m_ty - s_t * (m_ty); //m17
        m_tz = m_tz - s_t * (m_tz); //m18

        // transform back to PDFs
        // coeffcients for performance
        m_rho = mrt_coef1_d * m_rho;            //1 / 19
        m_e = mrt_coef2_d * m_e;              //1 / 2394
        m_e2 = mrt_coef3_d * m_e2;              //1 / 252
        m_jx = prc(0.1) * m_jx;
        m_qx = prc(0.025) * m_qx;
        m_jy = prc(0.1) * m_jy;
        m_qy = prc(0.025) * m_qy;
        m_jz = prc(0.1) * m_jz;
        m_qz = prc(0.025) * m_qz;
        m_3pxx = prc(2.) * mrt_coef4_d * m_3pxx;      //1 / 36
        m_3pixx = mrt_coef4_d * m_3pixx;        //1 / 72
        m_pww = prc(6.) * mrt_coef4_d * m_pww;       //1 / 12
        m_piww = prc(3.) * mrt_coef4_d * m_piww;     //1 / 24
        m_pxy = prc(0.25) * m_pxy;
        m_pyz = prc(0.25) * m_pyz;
        m_pzx = prc(0.25) * m_pzx;
        m_tx = prc(0.125) * m_tx;
        m_ty = prc(0.125) * m_ty;
        m_tz = prc(0.125) * m_tz;
        sum1 = m_rho - prc(11.) * m_e - prc(4.) * m_e2;
        sum2 = prc(2.) * m_3pxx - prc(4.) * m_3pixx;
        sum3 = m_pww - prc(2.) * m_piww;
        sum4 = m_rho + prc(8.) * m_e + m_e2;
        sum5 = m_jx + m_qx;
        sum6 = m_jy + m_qy;
        sum7 = m_jz + m_qz;
        sum8 = m_3pxx + m_3pixx;
        sum9 = m_pww + m_piww;

        ft0 = m_rho - prc(30.) * m_e + prc(12.) * m_e2;
        ft1 = sum1 + m_jx - prc(4.) * m_qx + sum2;
        ft2 = sum1 - m_jx + prc(4.) * m_qx + sum2;
        ft3 = sum1 + m_jy - prc(4.) * m_qy - prc(0.5) * sum2 + sum3;
        ft4 = sum1 - m_jy + prc(4.) * m_qy - prc(0.5) * sum2 + sum3;
        ft5 = sum1 + m_jz - prc(4.) * m_qz - prc(0.5) * sum2 - sum3;
        ft6 = sum1 - m_jz + prc(4.) * m_qz - prc(0.5) * sum2 - sum3;
        ft7 = sum4 + sum5 + sum6 + sum8 + sum9 + m_pxy + m_tx - m_ty;
        ft8 = sum4 - sum5 + sum6 + sum8 + sum9 - m_pxy - m_tx - m_ty;
        ft9 = sum4 + sum5 - sum6 + sum8 + sum9 - m_pxy + m_tx + m_ty;
        ft10 = sum4 - sum5 - sum6 + sum8 + sum9 + m_pxy - m_tx + m_ty;
        ft11 = sum4 + sum5 + sum7 + sum8 - sum9 + m_pzx - m_tx + m_tz;
        ft12 = sum4 - sum5 + sum7 + sum8 - sum9 - m_pzx + m_tx + m_tz;
        ft13 = sum4 + sum5 - sum7 + sum8 - sum9 - m_pzx - m_tx - m_tz;
        ft14 = sum4 - sum5 - sum7 + sum8 - sum9 + m_pzx + m_tx - m_tz;
        ft15 = sum4 + sum6 + sum7 - sum8 * prc(2.) + m_pyz + m_ty - m_tz;
        ft16 = sum4 - sum6 + sum7 - sum8 * prc(2.) - m_pyz - m_ty - m_tz;
        ft17 = sum4 + sum6 - sum7 - sum8 * prc(2.) - m_pyz + m_ty + m_tz;
        ft18 = sum4 - sum6 - sum7 - sum8 * prc(2.) + m_pyz - m_ty + m_tz;
        // ~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~MRT kernel, repeated part~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~

        // ++++++++++++ - recoloring & streaming to opposite direction++++++++++++++++++++++++++++++++++++++++++++++++++++++++++++++++++

        tmp1 = rho1 / den;
        pdf[i_f1(i, j, k, 0, 0)] = tmp1 * ft0;
        pdf[i_f1(i, j, k, 0, 1)] = ft0 * (prc(1.) - tmp1);

        // R - K method
        tmp = rho1 * rho2 * lbm_beta_d / den;

        g1t1 = tmp1 * ft1 + w_equ_1_d * tmp * (cnx);
        g1t2 = tmp1 * ft2 + w_equ_1_d * tmp * (-cnx);
        g1t3 = tmp1 * ft3 + w_equ_1_d * tmp * (cny);
        g1t4 = tmp1 * ft4 + w_equ_1_d * tmp * (-cny);
        g1t5 = tmp1 * ft5 + w_equ_1_d * tmp * (cnz);
        g1t6 = tmp1 * ft6 + w_equ_1_d * tmp * (-cnz);
        g1t7 = tmp1 * ft7 + RK_weight2_d * tmp * (cnx + cny);
        g1t8 = tmp1 * ft8 + RK_weight2_d * tmp * (-cnx + cny);
        g1t9 = tmp1 * ft9 + RK_weight2_d * tmp * (cnx - cny);
        g1t10 = tmp1 * ft10 + RK_weight2_d * tmp * (-cnx - cny);
        g1t11 = tmp1 * ft11 + RK_weight2_d * tmp * (cnx + cnz);
        g1t12 = tmp1 * ft12 + RK_weight2_d * tmp * (-cnx + cnz);
        g1t13 = tmp1 * ft13 + RK_weight2_d * tmp * (cnx - cnz);
        g1t14 = tmp1 * ft14 + RK_weight2_d * tmp * (-cnx - cnz);
        g1t15 = tmp1 * ft15 + RK_weight2_d * tmp * (cny + cnz);
        g1t16 = tmp1 * ft16 + RK_weight2_d * tmp * (-cny + cnz);
        g1t17 = tmp1 * ft17 + RK_weight2_d * tmp * (cny - cnz);
        g1t18 = tmp1 * ft18 + RK_weight2_d * tmp * (-cny - cnz);

        g2t1 = ft1 - g1t1;
        g2t2 = ft2 - g1t2;
        g2t3 = ft3 - g1t3;
        g2t4 = ft4 - g1t4;
        g2t5 = ft5 - g1t5;
        g2t6 = ft6 - g1t6;
        g2t7 = ft7 - g1t7;
        g2t8 = ft8 - g1t8;
        g2t9 = ft9 - g1t9;
        g2t10 = ft10 - g1t10;
        g2t11 = ft11 - g1t11;
        g2t12 = ft12 - g1t12;
        g2t13 = ft13 - g1t13;
        g2t14 = ft14 - g1t14;
        g2t15 = ft15 - g1t15;
        g2t16 = ft16 - g1t16;
        g2t17 = ft17 - g1t17;
        g2t18 = ft18 - g1t18;

        // ++++++++ + -AA pattern push step++++++++++++
        pdf[i_f1(i + 1, j, k, 2, 0)] = g1t1;
        pdf[i_f1(i - 1, j, k, 1, 0)] = g1t2;
        pdf[i_f1(i, j + 1, k, 4, 0)] = g1t3;
        pdf[i_f1(i, j - 1, k, 3, 0)] = g1t4;
        pdf[i_f1(i, j, k + 1, 6, 0)] = g1t5;
        pdf[i_f1(i, j, k - 1, 5, 0)] = g1t6;
        pdf[i_f1(i + 1, j + 1, k, 10, 0)] = g1t7;
        pdf[i_f1(i - 1, j + 1, k, 9, 0)] = g1t8;
        pdf[i_f1(i + 1, j - 1, k, 8, 0)] = g1t9;
        pdf[i_f1(i - 1, j - 1, k, 7, 0)] = g1t10;
        pdf[i_f1(i + 1, j, k + 1, 14, 0)] = g1t11;
        pdf[i_f1(i - 1, j, k + 1, 13, 0)] = g1t12;
        pdf[i_f1(i + 1, j, k - 1, 12, 0)] = g1t13;
        pdf[i_f1(i - 1, j, k - 1, 11, 0)] = g1t14;
        pdf[i_f1(i, j + 1, k + 1, 18, 0)] = g1t15;
        pdf[i_f1(i, j - 1, k + 1, 17, 0)] = g1t16;
        pdf[i_f1(i, j + 1, k - 1, 16, 0)] = g1t17;
        pdf[i_f1(i, j - 1, k - 1, 15, 0)] = g1t18;

        pdf[i_f1(i + 1, j, k, 2, 1)] = g2t1;
        pdf[i_f1(i - 1, j, k, 1, 1)] = g2t2;
        pdf[i_f1(i, j + 1, k, 4, 1)] = g2t3;
        pdf[i_f1(i, j - 1, k, 3, 1)] = g2t4;
        pdf[i_f1(i, j, k + 1, 6, 1)] = g2t5;
        pdf[i_f1(i, j, k - 1, 5, 1)] = g2t6;
        pdf[i_f1(i + 1, j + 1, k, 10, 1)] = g2t7;
        pdf[i_f1(i - 1, j + 1, k, 9, 1)] = g2t8;
        pdf[i_f1(i + 1, j - 1, k, 8, 1)] = g2t9;
        pdf[i_f1(i - 1, j - 1, k, 7, 1)] = g2t10;
        pdf[i_f1(i + 1, j, k + 1, 14, 1)] = g2t11;
        pdf[i_f1(i - 1, j, k + 1, 13, 1)] = g2t12;
        pdf[i_f1(i + 1, j, k - 1, 12, 1)] = g2t13;
        pdf[i_f1(i - 1, j, k - 1, 11, 1)] = g2t14;
        pdf[i_f1(i, j + 1, k + 1, 18, 1)] = g2t15;
        pdf[i_f1(i, j - 1, k + 1, 17, 1)] = g2t16;
        pdf[i_f1(i, j + 1, k - 1, 16, 1)] = g2t17;
        pdf[i_f1(i, j - 1, k - 1, 15, 1)] = g2t18;

    }

}

//=====================================================================================================================================
//----------------------even step kernel----------------------
//no steaming steps
//=====================================================================================================================================

__global__ void kernel_even_color_GPU(int ixmin, int ixmax, int iymin, int iymax, int izmin, int izmax,
    int* walls, T_P* pdf, T_P* phi, T_P* cn_x, T_P* cn_y, T_P* cn_z, T_P* curv, T_P* c_norm) {

    // Indexing (Thread)
    int i = threadIdx.x + blockIdx.x * blockDim.x + 1;
    int j = threadIdx.y + blockIdx.y * blockDim.y + 1;
    int k = threadIdx.z + blockIdx.z * blockDim.z + 1;

    T_P cnx, cny, cnz, rho1, rho2, tmp1;   //color model
    T_P sum1, sum2, sum3, sum4, sum5, sum6, sum7, sum8, sum9;
    T_P m_rho, m_e, m_e2, m_jx, m_qx, m_jy, m_qy, m_jz, m_qz, m_3pxx, m_3pixx, m_pww, m_piww, m_pxy, m_pyz, m_pzx, m_tx, m_ty, m_tz;
    T_P ft0, ft1, ft2, ft3, ft4, ft5, ft6, ft7, ft8, ft9, ft10, ft11, ft12, ft13, ft14, ft15, ft16, ft17, ft18;
    T_P g1t0, g1t1, g1t2, g1t3, g1t4, g1t5, g1t6, g1t7, g1t8, g1t9, g1t10, g1t11, g1t12, g1t13, g1t14, g1t15, g1t16, g1t17, g1t18;
    T_P g2t0, g2t1, g2t2, g2t3, g2t4, g2t5, g2t6, g2t7, g2t8, g2t9, g2t10, g2t11, g2t12, g2t13, g2t14, g2t15, g2t16, g2t17, g2t18;
    T_P s_e, s_e2, s_q, s_nu, s_pi, s_t;  //relaxation parameters
    T_P fx, fy, fz, ux1, uy1, uz1, den, tmp, omega, u2;

    if (walls[i_s2(i, j, k)] == 0 && i >= ixmin && i <= ixmax && j >= iymin && j <= iymax && k >= izmin && k <= izmax) {
        // +++++++++- AA pattern pull step++++++++++++
        g1t0 = pdf[i_f1(i, j, k, 0, 0)];
        g1t2 = pdf[i_f1(i, j, k, 1, 0)];
        g1t1 = pdf[i_f1(i, j, k, 2, 0)];
        g1t4 = pdf[i_f1(i, j, k, 3, 0)];
        g1t3 = pdf[i_f1(i, j, k, 4, 0)];
        g1t6 = pdf[i_f1(i, j, k, 5, 0)];
        g1t5 = pdf[i_f1(i, j, k, 6, 0)];
        g1t10 = pdf[i_f1(i, j, k, 7, 0)];
        g1t9 = pdf[i_f1(i, j, k, 8, 0)];
        g1t8 = pdf[i_f1(i, j, k, 9, 0)];
        g1t7 = pdf[i_f1(i, j, k, 10, 0)];
        g1t14 = pdf[i_f1(i, j, k, 11, 0)];
        g1t13 = pdf[i_f1(i, j, k, 12, 0)];
        g1t12 = pdf[i_f1(i, j, k, 13, 0)];
        g1t11 = pdf[i_f1(i, j, k, 14, 0)];
        g1t18 = pdf[i_f1(i, j, k, 15, 0)];
        g1t17 = pdf[i_f1(i, j, k, 16, 0)];
        g1t16 = pdf[i_f1(i, j, k, 17, 0)];
        g1t15 = pdf[i_f1(i, j, k, 18, 0)];

        g2t0 = pdf[i_f1(i, j, k, 0, 1)];
        g2t2 = pdf[i_f1(i, j, k, 1, 1)];
        g2t1 = pdf[i_f1(i, j, k, 2, 1)];
        g2t4 = pdf[i_f1(i, j, k, 3, 1)];
        g2t3 = pdf[i_f1(i, j, k, 4, 1)];
        g2t6 = pdf[i_f1(i, j, k, 5, 1)];
        g2t5 = pdf[i_f1(i, j, k, 6, 1)];
        g2t10 = pdf[i_f1(i, j, k, 7, 1)];
        g2t9 = pdf[i_f1(i, j, k, 8, 1)];
        g2t8 = pdf[i_f1(i, j, k, 9, 1)];
        g2t7 = pdf[i_f1(i, j, k, 10, 1)];
        g2t14 = pdf[i_f1(i, j, k, 11, 1)];
        g2t13 = pdf[i_f1(i, j, k, 12, 1)];
        g2t12 = pdf[i_f1(i, j, k, 13, 1)];
        g2t11 = pdf[i_f1(i, j, k, 14, 1)];
        g2t18 = pdf[i_f1(i, j, k, 15, 1)];
        g2t17 = pdf[i_f1(i, j, k, 16, 1)];
        g2t16 = pdf[i_f1(i, j, k, 17, 1)];
        g2t15 = pdf[i_f1(i, j, k, 18, 1)];

        // ft: bulk PDF
        ft0 = g1t0 + g2t0;
        ft1 = g1t1 + g2t1;
        ft2 = g1t2 + g2t2;
        ft3 = g1t3 + g2t3;
        ft4 = g1t4 + g2t4;
        ft5 = g1t5 + g2t5;
        ft6 = g1t6 + g2t6;
        ft7 = g1t7 + g2t7;
        ft8 = g1t8 + g2t8;
        ft9 = g1t9 + g2t9;
        ft10 = g1t10 + g2t10;
        ft11 = g1t11 + g2t11;
        ft12 = g1t12 + g2t12;
        ft13 = g1t13 + g2t13;
        ft14 = g1t14 + g2t14;
        ft15 = g1t15 + g2t15;
        ft16 = g1t16 + g2t16;
        ft17 = g1t17 + g2t17;
        ft18 = g1t18 + g2t18;

        // order parameter
        rho1 = g1t0 + g1t1 + g1t2 + g1t3 + g1t4 + g1t5 + g1t6 + g1t7 + g1t8 + g1t9 + g1t10 + g1t11 + g1t12 + g1t13 + g1t14 + g1t15 + g1t16 + g1t17 + g1t18;
        rho2 = g2t0 + g2t1 + g2t2 + g2t3 + g2t4 + g2t5 + g2t6 + g2t7 + g2t8 + g2t9 + g2t10 + g2t11 + g2t12 + g2t13 + g2t14 + g2t15 + g2t16 + g2t17 + g2t18;

        T_P phi_loc = (rho1 - rho2) / (rho1 + rho2);
        phi[i_s4(i, j, k)] = phi_loc;

        cnx = cn_x[i_s2(i, j, k)];
        cny = cn_y[i_s2(i, j, k)];
        cnz = cn_z[i_s2(i, j, k)];

        tmp = prc(0.5) * lbm_gamma_d * curv[i_s1(i, j, k)] * c_norm[i_s2(i, j, k)];
        fx = tmp * cnx;
        fy = tmp * cny;
        fz = tmp * cnz + force_z_d;   // body force force_z along flow direction

        // ++++++++++++ - MRT COLLISION++++++++++++ + -
        // select viscosity++++++++++++++++++++++ + -
        omega = prc(1.) / (prc(6.) / ((prc(1.0) + phi_loc) * la_nui1_d + (prc(1.0) - phi_loc) * la_nui2_d) + prc(0.5));
        // MRT PARAMETERS
        s_nu = omega;
#if (mrt==1)
        // ************bounceback opt * ***********
        s_e = omega;
        s_e2 = omega;
        s_pi = omega;
        s_q = prc(8.0) * (prc(2.0) - omega) / (prc(8.0) - omega);
        s_t = s_q;
#elif (mrt==2)
        //************original opt * ***********
        s_e = prc(1.19);
        s_e2 = prc(1.4);
        s_pi = prc(1.4);
        s_q = prc(1.2);
        s_t = prc(1.98);
#elif (mrt==3)
        // ************SRT * ***********
        s_e = omega;
        s_e2 = omega;
        s_pi = omega;
        s_q = omega;
        s_t = omega;
#elif (mrt==4)           
        // ************advection opt * ***********
        s_e = omega;
        s_e2 = omega;
        s_pi = omega;
        s_q = (prc(6.) - prc(3.) * omega) / (prc(3.) - omega);
        s_t = omega;
#endif

        // ~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~MRT kernel, repeated part~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~
        // ++++++++ + -!calculate macroscopic variables++++++++++++
        den = rho1 + rho2;

        ux1 = ft1 - ft2 + ft7 - ft8 + ft9 - ft10 + ft11 - ft12 + ft13 - ft14 + prc(0.5) * fx;
        uy1 = ft3 - ft4 + ft7 + ft8 - ft9 - ft10 + ft15 - ft16 + ft17 - ft18 + prc(0.5) * fy;
        uz1 = ft5 - ft6 + ft11 + ft12 - ft13 - ft14 + ft15 + ft16 - ft17 - ft18 + prc(0.5) * fz;
        u2 = ux1 * ux1 + uy1 * uy1 + uz1 * uz1;

        // PDFs summations for computation efficiency purpose
        sum1 = ft1 + ft2 + ft3 + ft4 + ft5 + ft6;
        sum2 = ft7 + ft8 + ft9 + ft10 + ft11 + ft12 + ft13 + ft14 + ft15 + ft16 + ft17 + ft18;
        sum3 = ft7 - ft8 + ft9 - ft10 + ft11 - ft12 + ft13 - ft14;
        sum4 = ft7 + ft8 - ft9 - ft10 + ft15 - ft16 + ft17 - ft18;
        sum5 = ft11 + ft12 - ft13 - ft14 + ft15 + ft16 - ft17 - ft18;
        sum6 = prc(2.) * (ft1 + ft2) - ft3 - ft4 - ft5 - ft6;
        sum7 = ft7 + ft8 + ft9 + ft10 + ft11 + ft12 + ft13 + ft14 - prc(2.) * (ft15 + ft16 + ft17 + ft18);
        sum8 = ft3 + ft4 - ft5 - ft6;
        sum9 = ft7 + ft8 + ft9 + ft10 - ft11 - ft12 - ft13 - ft14;
        // PDF to moment
        m_rho = den;
        m_e = prc(-30.) * ft0 - prc(11.) * sum1 + prc(8.) * sum2;
        m_e2 = prc(12.) * ft0 - prc(4.) * sum1 + sum2;
        m_jx = ft1 - ft2 + sum3;
        m_qx = prc(-4.) * (ft1 - ft2) + sum3;
        m_jy = ft3 - ft4 + sum4;
        m_qy = prc(-4.) * (ft3 - ft4) + sum4;
        m_jz = ft5 - ft6 + sum5;
        m_qz = prc(-4.) * (ft5 - ft6) + sum5;
        m_3pxx = sum6 + sum7;
        m_3pixx = prc(-2.) * sum6 + sum7;
        m_pww = sum8 + sum9;
        m_piww = prc(-2.) * sum8 + sum9;
        m_pxy = ft7 - ft8 - ft9 + ft10;
        m_pyz = ft15 - ft16 - ft17 + ft18;
        m_pzx = ft11 - ft12 - ft13 + ft14;
        m_tx = ft7 - ft8 + ft9 - ft10 - ft11 + ft12 - ft13 + ft14;
        m_ty = -ft7 - ft8 + ft9 + ft10 + ft15 - ft16 + ft17 - ft18;
        m_tz = ft11 + ft12 - ft13 - ft14 - ft15 - ft16 + ft17 + ft18;

        // relaxtion in moment space
        m_e = m_e - s_e * (m_e - (prc(-11.0) * den + prc(19.0) * u2)) + (prc(38.) - prc(19.) * s_e) * (fx * ux1 + fy * uy1 + fz * uz1);                           //m1
        m_e2 = m_e2 - s_e2 * (m_e2 - (mrt_e2_coef1_d * den + mrt_e2_coef2_d * u2)) + (prc(-11.) + prc(5.5) * s_e2) * (fx * ux1 + fy * uy1 + fz * uz1);                     //m2
        m_jx = m_jx + fx;                                                                                                  //m3
        m_qx = m_qx - s_q * (m_qx - (prc(-0.666666666666666667) * ux1)) + (prc(-0.666666666666666667) + prc(0.333333333333333333) * s_q) * fx; //m4
        m_jy = m_jy + fy;                                                                                                   //m5
        m_qy = m_qy - s_q * (m_qy - (prc(-0.666666666666666667) * uy1)) + (prc(-0.666666666666666667) + prc(0.333333333333333333) * s_q) * fy; //m6
        m_jz = m_jz + fz;                                                                                                  //m7
        m_qz = m_qz - s_q * (m_qz - (prc(-0.666666666666666667) * uz1)) + (prc(-0.666666666666666667) + prc(0.333333333333333333) * s_q) * fz; //m8

        m_3pxx = m_3pxx - s_nu * (m_3pxx - (prc(3.) * ux1 * ux1 - u2)) + (prc(2.) - s_nu) * (prc(2.) * fx * ux1 - fy * uy1 - fz * uz1);                            //m9
        m_3pixx = m_3pixx - s_pi * (m_3pixx - mrt_omega_xx_d * (prc(3.) * ux1 * ux1 - u2)) + (prc(1.) - prc(0.5) * s_pi) * (prc(-2.) * fx * ux1 + fy * uy1 + fz * uz1);         //m10
        m_pww = m_pww - s_nu * (m_pww - (uy1 * uy1 - uz1 * uz1)) + (prc(2.) - s_nu) * (fy * uy1 - fz * uz1);                                      //m11
        m_piww = m_piww - s_pi * (m_piww - mrt_omega_xx_d * (uy1 * uy1 - uz1 * uz1)) + (prc(1.) - prc(0.5) * s_pi) * (-fy * uy1 + fz * uz1);                   //m12
        m_pxy = m_pxy - s_nu * (m_pxy - (ux1 * uy1)) + (prc(1.) - prc(0.5) * s_nu) * (fx * uy1 + fy * ux1);                                        //m13
        m_pyz = m_pyz - s_nu * (m_pyz - (uy1 * uz1)) + (prc(1.) - prc(0.5) * s_nu) * (fy * uz1 + fz * uy1);                                        //m14
        m_pzx = m_pzx - s_nu * (m_pzx - (ux1 * uz1)) + (prc(1.) - prc(0.5) * s_nu) * (fx * uz1 + fz * ux1);                                        //m15
        m_tx = m_tx - s_t * (m_tx); //m16
        m_ty = m_ty - s_t * (m_ty); //m17
        m_tz = m_tz - s_t * (m_tz); //m18


        // transform back to PDFs
        // coeffcients for performance
        m_rho = mrt_coef1_d * m_rho;            //1 / 19
        m_e = mrt_coef2_d * m_e;              //1 / 2394
        m_e2 = mrt_coef3_d * m_e2;              //1 / 252
        m_jx = prc(0.1) * m_jx;
        m_qx = prc(0.025) * m_qx;
        m_jy = prc(0.1) * m_jy;
        m_qy = prc(0.025) * m_qy;
        m_jz = prc(0.1) * m_jz;
        m_qz = prc(0.025) * m_qz;
        m_3pxx = prc(2.) * mrt_coef4_d * m_3pxx;      //1 / 36
        m_3pixx = mrt_coef4_d * m_3pixx;          // 1 / 72
        m_pww = prc(6.) * mrt_coef4_d * m_pww;      //!1 / 12
        m_piww = prc(3.) * mrt_coef4_d * m_piww;     //1 / 24
        m_pxy = prc(0.25) * m_pxy;
        m_pyz = prc(0.25) * m_pyz;
        m_pzx = prc(0.25) * m_pzx;
        m_tx = prc(0.125) * m_tx;
        m_ty = prc(0.125) * m_ty;
        m_tz = prc(0.125) * m_tz;
        sum1 = m_rho - prc(11.) * m_e - prc(4.) * m_e2;
        sum2 = prc(2.) * m_3pxx - prc(4.) * m_3pixx;
        sum3 = m_pww - prc(2.) * m_piww;
        sum4 = m_rho + prc(8.) * m_e + m_e2;
        sum5 = m_jx + m_qx;
        sum6 = m_jy + m_qy;
        sum7 = m_jz + m_qz;
        sum8 = m_3pxx + m_3pixx;
        sum9 = m_pww + m_piww;

        ft0 = m_rho - prc(30.) * m_e + prc(12.) * m_e2;
        ft1 = sum1 + m_jx - prc(4.) * m_qx + sum2;
        ft2 = sum1 - m_jx + prc(4.) * m_qx + sum2;
        ft3 = sum1 + m_jy - prc(4.) * m_qy - prc(0.5) * sum2 + sum3;
        ft4 = sum1 - m_jy + prc(4.) * m_qy - prc(0.5) * sum2 + sum3;
        ft5 = sum1 + m_jz - prc(4.) * m_qz - prc(0.5) * sum2 - sum3;
        ft6 = sum1 - m_jz + prc(4.) * m_qz - prc(0.5) * sum2 - sum3;
        ft7 = sum4 + sum5 + sum6 + sum8 + sum9 + m_pxy + m_tx - m_ty;
        ft8 = sum4 - sum5 + sum6 + sum8 + sum9 - m_pxy - m_tx - m_ty;
        ft9 = sum4 + sum5 - sum6 + sum8 + sum9 - m_pxy + m_tx + m_ty;
        ft10 = sum4 - sum5 - sum6 + sum8 + sum9 + m_pxy - m_tx + m_ty;
        ft11 = sum4 + sum5 + sum7 + sum8 - sum9 + m_pzx - m_tx + m_tz;
        ft12 = sum4 - sum5 + sum7 + sum8 - sum9 - m_pzx + m_tx + m_tz;
        ft13 = sum4 + sum5 - sum7 + sum8 - sum9 - m_pzx - m_tx - m_tz;
        ft14 = sum4 - sum5 - sum7 + sum8 - sum9 + m_pzx + m_tx - m_tz;
        ft15 = sum4 + sum6 + sum7 - sum8 * prc(2.) + m_pyz + m_ty - m_tz;
        ft16 = sum4 - sum6 + sum7 - sum8 * prc(2.) - m_pyz - m_ty - m_tz;
        ft17 = sum4 + sum6 - sum7 - sum8 * prc(2.) - m_pyz + m_ty + m_tz;
        ft18 = sum4 - sum6 - sum7 - sum8 * prc(2.) + m_pyz - m_ty + m_tz;
        // ~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~MRT kernel, repeated part~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~

        // ++++++++++++ - recoloring++++++++++++++++++++++++++++++++++++++++++++++++++++++++++++++++++
        tmp1 = rho1 / den;
        pdf[i_f1(i, j, k, 0, 0)] = tmp1 * ft0;
        pdf[i_f1(i, j, k, 0, 1)] = ft0 * (prc(1.) - tmp1);

        // R - K method
        tmp = rho1 * rho2 * lbm_beta_d / den;

        g1t1 = tmp1 * ft1 + w_equ_1_d * tmp * (cnx);
        g1t2 = tmp1 * ft2 + w_equ_1_d * tmp * (-cnx);
        g1t3 = tmp1 * ft3 + w_equ_1_d * tmp * (cny);
        g1t4 = tmp1 * ft4 + w_equ_1_d * tmp * (-cny);
        g1t5 = tmp1 * ft5 + w_equ_1_d * tmp * (cnz);
        g1t6 = tmp1 * ft6 + w_equ_1_d * tmp * (-cnz);
        g1t7 = tmp1 * ft7 + RK_weight2_d * tmp * (cnx + cny);
        g1t8 = tmp1 * ft8 + RK_weight2_d * tmp * (-cnx + cny);
        g1t9 = tmp1 * ft9 + RK_weight2_d * tmp * (cnx - cny);
        g1t10 = tmp1 * ft10 + RK_weight2_d * tmp * (-cnx - cny);
        g1t11 = tmp1 * ft11 + RK_weight2_d * tmp * (cnx + cnz);
        g1t12 = tmp1 * ft12 + RK_weight2_d * tmp * (-cnx + cnz);
        g1t13 = tmp1 * ft13 + RK_weight2_d * tmp * (cnx - cnz);
        g1t14 = tmp1 * ft14 + RK_weight2_d * tmp * (-cnx - cnz);
        g1t15 = tmp1 * ft15 + RK_weight2_d * tmp * (cny + cnz);
        g1t16 = tmp1 * ft16 + RK_weight2_d * tmp * (-cny + cnz);
        g1t17 = tmp1 * ft17 + RK_weight2_d * tmp * (cny - cnz);
        g1t18 = tmp1 * ft18 + RK_weight2_d * tmp * (-cny - cnz);

        g2t1 = ft1 - g1t1;
        g2t2 = ft2 - g1t2;
        g2t3 = ft3 - g1t3;
        g2t4 = ft4 - g1t4;
        g2t5 = ft5 - g1t5;
        g2t6 = ft6 - g1t6;
        g2t7 = ft7 - g1t7;
        g2t8 = ft8 - g1t8;
        g2t9 = ft9 - g1t9;
        g2t10 = ft10 - g1t10;
        g2t11 = ft11 - g1t11;
        g2t12 = ft12 - g1t12;
        g2t13 = ft13 - g1t13;
        g2t14 = ft14 - g1t14;
        g2t15 = ft15 - g1t15;
        g2t16 = ft16 - g1t16;
        g2t17 = ft17 - g1t17;
        g2t18 = ft18 - g1t18;

        // ++++++++ + -AA pattern++++++++++++
        pdf[i_f1(i, j, k, 1, 0)] = g1t1;
        pdf[i_f1(i, j, k, 2, 0)] = g1t2;
        pdf[i_f1(i, j, k, 3, 0)] = g1t3;
        pdf[i_f1(i, j, k, 4, 0)] = g1t4;
        pdf[i_f1(i, j, k, 5, 0)] = g1t5;
        pdf[i_f1(i, j, k, 6, 0)] = g1t6;
        pdf[i_f1(i, j, k, 7, 0)] = g1t7;
        pdf[i_f1(i, j, k, 8, 0)] = g1t8;
        pdf[i_f1(i, j, k, 9, 0)] = g1t9;
        pdf[i_f1(i, j, k, 10, 0)] = g1t10;
        pdf[i_f1(i, j, k, 11, 0)] = g1t11;
        pdf[i_f1(i, j, k, 12, 0)] = g1t12;
        pdf[i_f1(i, j, k, 13, 0)] = g1t13;
        pdf[i_f1(i, j, k, 14, 0)] = g1t14;
        pdf[i_f1(i, j, k, 15, 0)] = g1t15;
        pdf[i_f1(i, j, k, 16, 0)] = g1t16;
        pdf[i_f1(i, j, k, 17, 0)] = g1t17;
        pdf[i_f1(i, j, k, 18, 0)] = g1t18;

        pdf[i_f1(i, j, k, 1, 1)] = g2t1;
        pdf[i_f1(i, j, k, 2, 1)] = g2t2;
        pdf[i_f1(i, j, k, 3, 1)] = g2t3;
        pdf[i_f1(i, j, k, 4, 1)] = g2t4;
        pdf[i_f1(i, j, k, 5, 1)] = g2t5;
        pdf[i_f1(i, j, k, 6, 1)] = g2t6;
        pdf[i_f1(i, j, k, 7, 1)] = g2t7;
        pdf[i_f1(i, j, k, 8, 1)] = g2t8;
        pdf[i_f1(i, j, k, 9, 1)] = g2t9;
        pdf[i_f1(i, j, k, 10, 1)] = g2t10;
        pdf[i_f1(i, j, k, 11, 1)] = g2t11;
        pdf[i_f1(i, j, k, 12, 1)] = g2t12;
        pdf[i_f1(i, j, k, 13, 1)] = g2t13;
        pdf[i_f1(i, j, k, 14, 1)] = g2t14;
        pdf[i_f1(i, j, k, 15, 1)] = g2t15;
        pdf[i_f1(i, j, k, 16, 1)] = g2t16;
        pdf[i_f1(i, j, k, 17, 1)] = g2t17;
        pdf[i_f1(i, j, k, 18, 1)] = g2t18;
    }

}

#pragma endregion (kernel_multiphase)

#pragma region (color gradient)
/* ~~~~~~~~~~~~~~~~~~~~~~~ extrapolate phi values to solid boundary nodes ~~~~~~~~~~~~~~~~~~ */
__global__ void extrapolate_phi_toSolid(int* walls_type, T_P* phi) {
    int i = threadIdx.x + blockIdx.x * blockDim.x + 1 - 3;
    int j = threadIdx.y + blockIdx.y * blockDim.y + 1 - 3;
    int k = threadIdx.z + blockIdx.z * blockDim.z + 1 - 3;

    if (i <= nxGlobal_d + 3 && j<= nyGlobal_d + 3 && k <= nzGlobal_d + 3) {
        int node_type_loc = walls_type[i_s4(i, j, k)];
        if (node_type_loc == 2) { // solid boundary node
            T_P phi_sum = prc(0.), weight_sum = prc(0.);
            for (int q = 1; q < 19; q++) {
                int iex = i + ex_d[q];
                int iey = j + ey_d[q];
                int iez = k + ez_d[q];
                int node_type_neb = walls_type[i_s4(iex, iey, iez)];
                if (node_type_neb <= 0) {
                    phi_sum += phi[i_s4(iex, iey, iez)] * w_equ_d[q];
                    weight_sum += w_equ_d[q];
                }
            }
            phi[i_s4(i, j, k)] = phi_sum / weight_sum;
        }
    }

}
/* ~~~~~~~~~~~~~~~~~~ calculate normal directions of interfaces from phi gradient ~~~~~~~~~~~~~~~~~~ */
__global__ void normalDirectionsOfInterfaces(int* walls, T_P* phi, T_P* cn_x, T_P* cn_y, T_P* cn_z, T_P* c_norm) {
    //int overlap_temp = 2;
    // Indexing (Thread)
    int i = threadIdx.x + blockIdx.x * blockDim.x + 1 - 2;
    int j = threadIdx.y + blockIdx.y * blockDim.y + 1 - 2;
    int k = threadIdx.z + blockIdx.z * blockDim.z + 1 - 2;

    if (i < nxGlobal_d + 4 && j < nyGlobal_d + 4 && k < nzGlobal_d + 4) {
        cn_x[i_s2(i, j, k)] =
            ISO4_d[1 - 1] * (phi[i_s4(i + 1, j, k)] - phi[i_s4(i - 1, j, k)]) +

            ISO4_d[2 - 1] * (
                phi[i_s4(i + 1, j + 1, k)] - phi[i_s4(i - 1, j - 1, k)] +
                phi[i_s4(i + 1, j - 1, k)] - phi[i_s4(i - 1, j + 1, k)] +
                phi[i_s4(i + 1, j, k + 1)] - phi[i_s4(i - 1, j, k - 1)] +
                phi[i_s4(i + 1, j, k - 1)] - phi[i_s4(i - 1, j, k + 1)]);

        cn_y[i_s2(i, j, k)] =
            ISO4_d[1 - 1] * (phi[i_s4(i, j + 1, k)] - phi[i_s4(i, j - 1, k)]) +

            ISO4_d[2 - 1] * (
                phi[i_s4(i + 1, j + 1, k)] - phi[i_s4(i - 1, j - 1, k)] +
                phi[i_s4(i - 1, j + 1, k)] - phi[i_s4(i + 1, j - 1, k)] +
                phi[i_s4(i, j + 1, k + 1)] - phi[i_s4(i, j - 1, k - 1)] +
                phi[i_s4(i, j + 1, k - 1)] - phi[i_s4(i, j - 1, k + 1)]);


        cn_z[i_s2(i, j, k)] =
            ISO4_d[1 - 1] * (phi[i_s4(i, j, k + 1)] - phi[i_s4(i, j, k - 1)]) +

            ISO4_d[2 - 1] * (
                phi[i_s4(i + 1, j, k + 1)] - phi[i_s4(i - 1, j, k - 1)] +
                phi[i_s4(i - 1, j, k + 1)] - phi[i_s4(i + 1, j, k - 1)] +
                phi[i_s4(i, j + 1, k + 1)] - phi[i_s4(i, j - 1, k - 1)] +
                phi[i_s4(i, j - 1, k + 1)] - phi[i_s4(i, j + 1, k - 1)]);

        c_norm[i_s2(i, j, k)] = prc(sqrt)(cn_x[i_s2(i, j, k)] * cn_x[i_s2(i, j, k)] + cn_y[i_s2(i, j, k)] * cn_y[i_s2(i, j, k)] + cn_z[i_s2(i, j, k)] * cn_z[i_s2(i, j, k)]);

        if (c_norm[i_s2(i, j, k)] < prc(1e-6) || walls[i_s2(i, j, k)] == 1) {
            cn_x[i_s2(i, j, k)] = prc(0.);
            cn_y[i_s2(i, j, k)] = prc(0.);
            cn_z[i_s2(i, j, k)] = prc(0.);
            c_norm[i_s2(i, j, k)] = prc(0.);
        }
        else {
            cn_x[i_s2(i, j, k)] = cn_x[i_s2(i, j, k)] / c_norm[i_s2(i, j, k)];
            cn_y[i_s2(i, j, k)] = cn_y[i_s2(i, j, k)] / c_norm[i_s2(i, j, k)];
            cn_z[i_s2(i, j, k)] = cn_z[i_s2(i, j, k)] / c_norm[i_s2(i, j, k)];    //normalized color gradient - interface normal direction
        }
    }
}
/* ~~~~~~~~~~~~~~ extrapolate normal direction info to solid boundary nodes, to minimize unbalanced forces ~~~~~~~~~~~~~~ */
__global__ void alter_color_gradient_solid_surface_GPU(int* walls_type, T_P* cn_x, T_P* cn_y, T_P* cn_z, T_P* c_norm, T_P* s_nx, T_P* s_ny, T_P* s_nz) {
    int i = threadIdx.x + blockIdx.x * blockDim.x + 1 - 2;
    int j = threadIdx.y + blockIdx.y * blockDim.y + 1 - 2;
    int k = threadIdx.z + blockIdx.z * blockDim.z + 1 - 2;

    if (i <= nxGlobal_d + 2 && j <= nyGlobal_d + 2 && k <= nzGlobal_d + 2) {
        int node_type_loc = walls_type[i_s4(i, j, k)];
        if (node_type_loc == -1) { // fluid boundary node
            int iteration, iteration_max;
            T_P nwx, nwy, nwz, lambda, local_eps;
            T_P vcx0, vcy0, vcz0, vcx1, vcy1, vcz1, vcx2, vcy2, vcz2, err0, err1, err2, tmp;

            lambda = prc(0.5);
            local_eps = prc(1e-6);
            iteration_max = 4;

            if (c_norm[i_s2(i, j, k)] > local_eps) {

                nwx = s_nx[i_s4(i, j, k)];
                nwy = s_ny[i_s4(i, j, k)];
                nwz = s_nz[i_s4(i, j, k)];
                vcx0 = cn_x[i_s2(i, j, k)];
                vcy0 = cn_y[i_s2(i, j, k)];
                vcz0 = cn_z[i_s2(i, j, k)];
                vcx1 = vcx0 - lambda * (vcx0 + nwx);
                vcy1 = vcy0 - lambda * (vcy0 + nwy);
                vcz1 = vcz0 - lambda * (vcz0 + nwz);

                err0 = (nwx * vcx0 + nwy * vcy0 + nwz * vcz0) - cos_theta_d;

                if ((pprc(abs)(vcx0 + nwx) + pprc(abs)(vcy0 + nwy) + pprc(abs)(vcz0 + nwz) > local_eps || pprc(abs)(vcx0 - nwx) + pprc(abs)(vcy0 - nwy) + pprc(abs)(vcz0 - nwz) > local_eps) && err0 > local_eps) {
                    // do not perform alteration when the normal direction of the solid surface aligned with the fluid interface direction,
                    // or the initial fluid direction is already the desired direction
                    err1 = (nwx * vcx1 + nwy * vcy1 + nwz * vcz1) - prc(sqrt)(vcx1 * vcx1 + vcy1 * vcy1 + vcz1 * vcz1) * cos_theta_d;
                    tmp = prc(1.) / (err1 - err0);
                    vcx2 = tmp * (vcx0 * err1 - vcx1 * err0);
                    vcy2 = tmp * (vcy0 * err1 - vcy1 * err0);
                    vcz2 = tmp * (vcz0 * err1 - vcz1 * err0);

                    err2 = (nwx * vcx2 + nwy * vcy2 + nwz * vcz2) - prc(sqrt)(vcx2 * vcx2 + vcy2 * vcy2 + vcz2 * vcz2) * cos_theta_d;

                    if (err2 > local_eps) {
                        for (iteration = 2; iteration <= iteration_max; iteration++) {
                            vcx0 = vcx1;
                            vcy0 = vcy1;
                            vcz0 = vcz1;
                            vcx1 = vcx2;
                            vcy1 = vcy2;
                            vcz1 = vcz2;
                            err0 = (nwx * vcx0 + nwy * vcy0 + nwz * vcz0) - prc(sqrt)(vcx0 * vcx0 + vcy0 * vcy0 + vcz0 * vcz0) * cos_theta_d;
                            err1 = (nwx * vcx1 + nwy * vcy1 + nwz * vcz1) - prc(sqrt)(vcx1 * vcx1 + vcy1 * vcy1 + vcz1 * vcz1) * cos_theta_d;
                            tmp = prc(1.) / (err1 - err0);
                            if (isinf(tmp)) break;
                            vcx2 = tmp * (vcx0 * err1 - vcx1 * err0);
                            vcy2 = tmp * (vcy0 * err1 - vcy1 * err0);
                            vcz2 = tmp * (vcz0 * err1 - vcz1 * err0);
                            err2 = (nwx * vcx2 + nwy * vcy2 + nwz * vcz2) - prc(sqrt)(vcx2 * vcx2 + vcy2 * vcy2 + vcz2 * vcz2) * cos_theta_d;
                        }
                        //if (iteration >= iteration_max)print*, 'after', iteration_max, ' iterations, theta=', dacos((nwx * vcx2 + nwy * vcy2 + nwz * vcz2) / dsqrt(vcx2 * vcx2 + vcy2 * vcy2 + vcz2 * vcz2)) / pi * 180
                    }
                    tmp = prc(1.) / ((prc(1e-30)) + prc(sqrt)(vcx2 * vcx2 + vcy2 * vcy2 + vcz2 * vcz2));
                    cn_x[i_s2(i, j, k)] = vcx2 * tmp;
                    cn_y[i_s2(i, j, k)] = vcy2 * tmp;
                    cn_z[i_s2(i, j, k)] = vcz2 * tmp;
                }
            }
        }
    }

}
/* ~~~~~~~~~~~~~~ extrapolate normal direction info to solid boundary nodes, to minimize unbalanced forces ~~~~~~~~~~~~~~ */
__global__ void extrapolateNormalToSolid(int* walls_type, T_P* cn_x, T_P* cn_y, T_P* cn_z, T_P* phi) {
    int i = threadIdx.x + blockIdx.x * blockDim.x + 1 - 1;
    int j = threadIdx.y + blockIdx.y * blockDim.y + 1 - 1;
    int k = threadIdx.z + blockIdx.z * blockDim.z + 1 - 1;

    if (i <= nxGlobal_d + 1 && j <= nyGlobal_d + 1 && k <= nzGlobal_d + 1) {
        int node_type_loc = walls_type[i_s4(i, j, k)];
        if (node_type_loc == 2) { // solid boundary node
            T_P cn_x_sum = prc(0.), cn_y_sum = prc(0.), cn_z_sum = prc(0.), weight_sum = prc(0.);
            for (int q = 1; q < 19; q++) {
                int iex = i + ex_d[q];
                int iey = j + ey_d[q];
                int iez = k + ez_d[q];
                int node_type_neb = walls_type[i_s4(iex, iey, iez)];
                if (node_type_neb <= 0) {
                    cn_x_sum += cn_x[i_s2(iex, iey, iez)] * w_equ_d[q];
                    cn_y_sum += cn_y[i_s2(iex, iey, iez)] * w_equ_d[q];
                    cn_z_sum += cn_z[i_s2(iex, iey, iez)] * w_equ_d[q];
                    weight_sum += w_equ_d[q];
                }
            }
            cn_x[i_s2(i, j, k)] = cn_x_sum / weight_sum;
            cn_y[i_s2(i, j, k)] = cn_y_sum / weight_sum;
            cn_z[i_s2(i, j, k)] = cn_z_sum / weight_sum;
        }
    }
}
/* ~~~~~~~~~~~~~~~~~~ calculate CSF forces based on interace curvature  ~~~~~~~~~~~~~~~~~~ */
__global__ void CSF_Forces(T_P* cn_x, T_P* cn_y, T_P* cn_z, T_P* curv) {
    // Indexing (Thread)
    int i = threadIdx.x + blockIdx.x * blockDim.x + 1;
    int j = threadIdx.y + blockIdx.y * blockDim.y + 1;
    int k = threadIdx.z + blockIdx.z * blockDim.z + 1;

    T_P kxx, kxy, kxz, kyx, kyy, kyz, kzx, kzy, kzz;
    if (i <= nxGlobal_d && j <= nyGlobal_d && k <= nzGlobal_d) {
        kxx =
            ISO4_d[1 - 1] * (cn_x[i_s2(i + 1, j, k)] - cn_x[i_s2(i - 1, j, k)]) +

            ISO4_d[2 - 1] * (
                cn_x[i_s2(i + 1, j + 1, k)] - cn_x[i_s2(i - 1, j - 1, k)] +
                cn_x[i_s2(i + 1, j - 1, k)] - cn_x[i_s2(i - 1, j + 1, k)] +
                cn_x[i_s2(i + 1, j, k + 1)] - cn_x[i_s2(i - 1, j, k - 1)] +
                cn_x[i_s2(i + 1, j, k - 1)] - cn_x[i_s2(i - 1, j, k + 1)]);

        kyy =
            ISO4_d[1 - 1] * (cn_y[i_s2(i, j + 1, k)] - cn_y[i_s2(i, j - 1, k)]) +

            ISO4_d[2 - 1] * (
                cn_y[i_s2(i + 1, j + 1, k)] - cn_y[i_s2(i - 1, j - 1, k)] +
                cn_y[i_s2(i - 1, j + 1, k)] - cn_y[i_s2(i + 1, j - 1, k)] +
                cn_y[i_s2(i, j + 1, k + 1)] - cn_y[i_s2(i, j - 1, k - 1)] +
                cn_y[i_s2(i, j + 1, k - 1)] - cn_y[i_s2(i, j - 1, k + 1)]);

        kzz =
            ISO4_d[1 - 1] * (cn_z[i_s2(i, j, k + 1)] - cn_z[i_s2(i, j, k - 1)]) +

            ISO4_d[2 - 1] * (
                cn_z[i_s2(i + 1, j, k + 1)] - cn_z[i_s2(i - 1, j, k - 1)] +
                cn_z[i_s2(i - 1, j, k + 1)] - cn_z[i_s2(i + 1, j, k - 1)] +
                cn_z[i_s2(i, j + 1, k + 1)] - cn_z[i_s2(i, j - 1, k - 1)] +
                cn_z[i_s2(i, j - 1, k + 1)] - cn_z[i_s2(i, j + 1, k - 1)]);


        kxy =
            ISO4_d[1 - 1] * (cn_x[i_s2(i, j + 1, k)] - cn_x[i_s2(i, j - 1, k)]) +

            ISO4_d[2 - 1] * (
                cn_x[i_s2(i + 1, j + 1, k)] - cn_x[i_s2(i - 1, j - 1, k)] +
                cn_x[i_s2(i - 1, j + 1, k)] - cn_x[i_s2(i + 1, j - 1, k)] +
                cn_x[i_s2(i, j + 1, k + 1)] - cn_x[i_s2(i, j - 1, k - 1)] +
                cn_x[i_s2(i, j + 1, k - 1)] - cn_x[i_s2(i, j - 1, k + 1)]);

        kxz =
            ISO4_d[1 - 1] * (cn_x[i_s2(i, j, k + 1)] - cn_x[i_s2(i, j, k - 1)]) +

            ISO4_d[2 - 1] * (
                cn_x[i_s2(i + 1, j, k + 1)] - cn_x[i_s2(i - 1, j, k - 1)] +
                cn_x[i_s2(i - 1, j, k + 1)] - cn_x[i_s2(i + 1, j, k - 1)] +
                cn_x[i_s2(i, j + 1, k + 1)] - cn_x[i_s2(i, j - 1, k - 1)] +
                cn_x[i_s2(i, j - 1, k + 1)] - cn_x[i_s2(i, j + 1, k - 1)]);

        kyx =
            ISO4_d[1 - 1] * (cn_y[i_s2(i + 1, j, k)] - cn_y[i_s2(i - 1, j, k)]) +

            ISO4_d[2 - 1] * (
                cn_y[i_s2(i + 1, j + 1, k)] - cn_y[i_s2(i - 1, j - 1, k)] +
                cn_y[i_s2(i + 1, j - 1, k)] - cn_y[i_s2(i - 1, j + 1, k)] +
                cn_y[i_s2(i + 1, j, k + 1)] - cn_y[i_s2(i - 1, j, k - 1)] +
                cn_y[i_s2(i + 1, j, k - 1)] - cn_y[i_s2(i - 1, j, k + 1)]);

        kyz =
            ISO4_d[1 - 1] * (cn_y[i_s2(i, j, k + 1)] - cn_y[i_s2(i, j, k - 1)]) +

            ISO4_d[2 - 1] * (
                cn_y[i_s2(i + 1, j, k + 1)] - cn_y[i_s2(i - 1, j, k - 1)] +
                cn_y[i_s2(i - 1, j, k + 1)] - cn_y[i_s2(i + 1, j, k - 1)] +
                cn_y[i_s2(i, j + 1, k + 1)] - cn_y[i_s2(i, j - 1, k - 1)] +
                cn_y[i_s2(i, j - 1, k + 1)] - cn_y[i_s2(i, j + 1, k - 1)]);

        kzx =
            ISO4_d[1 - 1] * (cn_z[i_s2(i + 1, j, k)] - cn_z[i_s2(i - 1, j, k)]) +

            ISO4_d[2 - 1] * (
                cn_z[i_s2(i + 1, j + 1, k)] - cn_z[i_s2(i - 1, j - 1, k)] +
                cn_z[i_s2(i + 1, j - 1, k)] - cn_z[i_s2(i - 1, j + 1, k)] +
                cn_z[i_s2(i + 1, j, k + 1)] - cn_z[i_s2(i - 1, j, k - 1)] +
                cn_z[i_s2(i + 1, j, k - 1)] - cn_z[i_s2(i - 1, j, k + 1)]);

        kzy =
            ISO4_d[1 - 1] * (cn_z[i_s2(i, j + 1, k)] - cn_z[i_s2(i, j - 1, k)]) +

            ISO4_d[2 - 1] * (
                cn_z[i_s2(i + 1, j + 1, k)] - cn_z[i_s2(i - 1, j - 1, k)] +
                cn_z[i_s2(i - 1, j + 1, k)] - cn_z[i_s2(i + 1, j - 1, k)] +
                cn_z[i_s2(i, j + 1, k + 1)] - cn_z[i_s2(i, j - 1, k - 1)] +
                cn_z[i_s2(i, j + 1, k - 1)] - cn_z[i_s2(i, j - 1, k + 1)]);

        curv[i_s1(i, j, k)] = (prc(pow)(cn_x[i_s2(i, j, k)], 2) - prc(1.)) * kxx + (prc(pow)(cn_y[i_s2(i, j, k)], 2) - prc(1.)) * kyy
            + (prc(pow)(cn_z[i_s2(i, j, k)], 2) - prc(1.)) * kzz +
            cn_x[i_s2(i, j, k)] * cn_y[i_s2(i, j, k)] * (kxy + kyx) + cn_x[i_s2(i, j, k)] * cn_z[i_s2(i, j, k)] * (kxz + kzx)
            + cn_y[i_s2(i, j, k)] * cn_z[i_s2(i, j, k)] * (kzy + kyz);
    }
}
#pragma endregion (color gradient)

#pragma region (Boundary Conditions)

#pragma region (Inlet Boundary Conditions)
__global__ void inlet_bounce_back_velocity_BC_before_odd_GPU(int* walls, T_P* phi, T_P* pdf, T_P* W_in) {

    int i = threadIdx.x + blockIdx.x * blockDim.x + 1;
    int j = threadIdx.y + blockIdx.y * blockDim.y + 1;

    if (i <= nxGlobal_d && j <= nyGlobal_d) {

        int wall_indicator;
        T_P tmp1, tmp2;

        wall_indicator = walls[i_s2(i, j, 1)];

        phi[i_s4(i, j, 0)] = phi_inlet_d * (1 - wall_indicator) + phi[i_s4(i, j, 0)] * wall_indicator;
        phi[i_s4(i, j, -1)] = phi[i_s4(i, j, 0)];
        phi[i_s4(i, j, -2)] = phi[i_s4(i, j, 0)];
        phi[i_s4(i, j, -3)] = phi[i_s4(i, j, 0)];   //overlap_phi = 4

        //inlet velocity BC    k = 1  ~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~
        tmp2 = W_in[i_s1(i, j, 0)] * relaxation_d;
        tmp1 = tmp2 * sa_inject_d;              //fluid 1 injection
        tmp2 = tmp2 - tmp1;                   //fluid 2 injection

        pdf[i_f1(i, j, 0, 5, 0)] = (pdf[i_f1(i, j, 1, 6, 0)] + prc(6.0) * w_equ_1_d * tmp1) * (1 - wall_indicator) + pdf[i_f1(i, j, 0, 5, 0)] * wall_indicator;
        pdf[i_f1(i - 1, j, 0, 11, 0)] = (pdf[i_f1(i, j, 1, 14, 0)] + prc(6.0) * w_equ_2_d * tmp1) * (1 - wall_indicator) + pdf[i_f1(i - 1, j, 0, 11, 0)] * wall_indicator;
        pdf[i_f1(i + 1, j, 0, 12, 0)] = (pdf[i_f1(i, j, 1, 13, 0)] + prc(6.0) * w_equ_2_d * tmp1) * (1 - wall_indicator) + pdf[i_f1(i + 1, j, 0, 12, 0)] * wall_indicator;
        pdf[i_f1(i, j - 1, 0, 15, 0)] = (pdf[i_f1(i, j, 1, 18, 0)] + prc(6.0) * w_equ_2_d * tmp1) * (1 - wall_indicator) + pdf[i_f1(i, j - 1, 0, 15, 0)] * wall_indicator;
        pdf[i_f1(i, j + 1, 0, 16, 0)] = (pdf[i_f1(i, j, 1, 17, 0)] + prc(6.0) * w_equ_2_d * tmp1) * (1 - wall_indicator) + pdf[i_f1(i, j + 1, 0, 16, 0)] * wall_indicator;

        pdf[i_f1(i, j, 0, 5, 1)] = (pdf[i_f1(i, j, 1, 6, 1)] + prc(6.0) * w_equ_1_d * tmp2) * (1 - wall_indicator) + pdf[i_f1(i, j, 0, 5, 1)] * wall_indicator;
        pdf[i_f1(i - 1, j, 0, 11, 1)] = (pdf[i_f1(i, j, 1, 14, 1)] + prc(6.0) * w_equ_2_d * tmp2) * (1 - wall_indicator) + pdf[i_f1(i - 1, j, 0, 11, 1)] * wall_indicator;
        pdf[i_f1(i + 1, j, 0, 12, 1)] = (pdf[i_f1(i, j, 1, 13, 1)] + prc(6.0) * w_equ_2_d * tmp2) * (1 - wall_indicator) + pdf[i_f1(i + 1, j, 0, 12, 1)] * wall_indicator;
        pdf[i_f1(i, j - 1, 0, 15, 1)] = (pdf[i_f1(i, j, 1, 18, 1)] + prc(6.0) * w_equ_2_d * tmp2) * (1 - wall_indicator) + pdf[i_f1(i, j - 1, 0, 15, 1)] * wall_indicator;
        pdf[i_f1(i, j + 1, 0, 16, 1)] = (pdf[i_f1(i, j, 1, 17, 1)] + prc(6.0) * w_equ_2_d * tmp2) * (1 - wall_indicator) + pdf[i_f1(i, j + 1, 0, 16, 1)] * wall_indicator;
    }
}

__global__ void inlet_bounce_back_velocity_BC_after_odd_GPU(int* walls, T_P* phi, T_P* pdf, T_P* W_in) {

    int i = threadIdx.x + blockIdx.x * blockDim.x + 1;
    int j = threadIdx.y + blockIdx.y * blockDim.y + 1;
    if (i <= nxGlobal_d && j <= nyGlobal_d) {
        int wall_indicator;
        T_P tmp1, tmp2;

        wall_indicator = walls[i_s2(i, j, 1)];

        phi[i_s4(i, j, 0)] = phi_inlet_d * (1 - wall_indicator) + phi[i_s4(i, j, 0)] * wall_indicator;
        phi[i_s4(i, j, -1)] = phi[i_s4(i, j, 0)];
        phi[i_s4(i, j, -2)] = phi[i_s4(i, j, 0)];
        phi[i_s4(i, j, -3)] = phi[i_s4(i, j, 0)];   //overlap_phi = 4

        // inlet velocity BC    k = 1  ~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~
        tmp2 = W_in[i_s1(i, j, 0)] * relaxation_d;
        tmp1 = tmp2 * sa_inject_d;              //fluid 1 injection
        tmp2 = tmp2 - tmp1;                   //fluid 2 injection

        pdf[i_f1(i, j, 1, 6, 0)] = (pdf[i_f1(i, j, 0, 5, 0)] + prc(6.0) * w_equ_1_d * tmp1) * (1 - wall_indicator) + pdf[i_f1(i, j, 1, 6, 0)] * wall_indicator;
        pdf[i_f1(i, j, 1, 13, 0)] = (pdf[i_f1(i + 1, j, 0, 12, 0)] + prc(6.0) * w_equ_2_d * tmp1) * (1 - wall_indicator) + pdf[i_f1(i, j, 1, 13, 0)] * wall_indicator;
        pdf[i_f1(i, j, 1, 14, 0)] = (pdf[i_f1(i - 1, j, 0, 11, 0)] + prc(6.0) * w_equ_2_d * tmp1) * (1 - wall_indicator) + pdf[i_f1(i, j, 1, 14, 0)] * wall_indicator;
        pdf[i_f1(i, j, 1, 17, 0)] = (pdf[i_f1(i, j + 1, 0, 16, 0)] + prc(6.0) * w_equ_2_d * tmp1) * (1 - wall_indicator) + pdf[i_f1(i, j, 1, 17, 0)] * wall_indicator;
        pdf[i_f1(i, j, 1, 18, 0)] = (pdf[i_f1(i, j - 1, 0, 15, 0)] + prc(6.0) * w_equ_2_d * tmp1) * (1 - wall_indicator) + pdf[i_f1(i, j, 1, 18, 0)] * wall_indicator;

        pdf[i_f1(i, j, 1, 6, 1)] = (pdf[i_f1(i, j, 0, 5, 1)] + prc(6.0) * w_equ_1_d * tmp2) * (1 - wall_indicator) + pdf[i_f1(i, j, 1, 6, 1)] * wall_indicator;
        pdf[i_f1(i, j, 1, 13, 1)] = (pdf[i_f1(i + 1, j, 0, 12, 1)] + prc(6.0) * w_equ_2_d * tmp2) * (1 - wall_indicator) + pdf[i_f1(i, j, 1, 13, 1)] * wall_indicator;
        pdf[i_f1(i, j, 1, 14, 1)] = (pdf[i_f1(i - 1, j, 0, 11, 1)] + prc(6.0) * w_equ_2_d * tmp2) * (1 - wall_indicator) + pdf[i_f1(i, j, 1, 14, 1)] * wall_indicator;
        pdf[i_f1(i, j, 1, 17, 1)] = (pdf[i_f1(i, j + 1, 0, 16, 1)] + prc(6.0) * w_equ_2_d * tmp2) * (1 - wall_indicator) + pdf[i_f1(i, j, 1, 17, 1)] * wall_indicator;
        pdf[i_f1(i, j, 1, 18, 1)] = (pdf[i_f1(i, j - 1, 0, 15, 1)] + prc(6.0) * w_equ_2_d * tmp2) * (1 - wall_indicator) + pdf[i_f1(i, j, 1, 18, 1)] * wall_indicator;
    }
}

//================================================================================================================================================================= =
//----------------------Zou - He type pressure / velocity open inlet boundary conditions----------------------
//currently, there should be only one dominant phase at the inlet boundary nodes,
//otherwise recolor scheme conflicts with zou - he BC for individual fluid component(momentumn for individual fluid is not conserved)
//================================================================================================================================================================= =
//**************************before odd step kernel * ****************************************
__global__ void inlet_Zou_He_pressure_BC_before_odd_GPU(T_P rho_in, int* walls, T_P* phi, T_P* pdf) {

    int i = threadIdx.x + blockIdx.x * blockDim.x + 1;
    int j = threadIdx.y + blockIdx.y * blockDim.y + 1;
    if (i <= nxGlobal_d && j <= nyGlobal_d) {
        int wall_indicator;

        T_P tmp1, tmp2, tnx, tny, tmpRho1, tmpRho2;

        wall_indicator = walls[i_s2(i, j, 1)];

        phi[i_s4(i, j, 0)] = phi_inlet_d * (1 - wall_indicator) + phi[i_s4(i, j, 0)] * wall_indicator;
        phi[i_s4(i, j, -1)] = phi[i_s4(i, j, 0)];
        phi[i_s4(i, j, -2)] = phi[i_s4(i, j, 0)];
        phi[i_s4(i, j, -3)] = phi[i_s4(i, j, 0)];   //overlap_phi = 4

        // inlet pressure BC    k = 1  ~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~
        // Zou - He pressure BC applied to the bulk PDF
        // inlet velocity BC    k = 1  ~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~
        tmpRho2 = rho_in;
        tmpRho1 = rho_in * sa_inject_d;                  //fluid 1 injection
        tmpRho2 = tmpRho2 - tmpRho1;                   // fluid 2 injection

        tmp1 = (tmpRho1 -
            (pdf[i_f1(i, j, 1, 0, 0)] +
                pdf[i_f1(i - 1, j, 1, 1, 0)] +
                pdf[i_f1(i + 1, j, 1, 2, 0)] +
                pdf[i_f1(i, j - 1, 1, 3, 0)] +
                pdf[i_f1(i, j + 1, 1, 4, 0)] +
                pdf[i_f1(i - 1, j - 1, 1, 7, 0)] +
                pdf[i_f1(i + 1, j - 1, 1, 8, 0)] +
                pdf[i_f1(i - 1, j + 1, 1, 9, 0)] +
                pdf[i_f1(i + 1, j + 1, 1, 10, 0)] + prc(2.) * (
                    pdf[i_f1(i, j, 2, 6, 0)] +
                    pdf[i_f1(i + 1, j, 2, 14, 0)] +
                    pdf[i_f1(i - 1, j, 2, 13, 0)] +
                    pdf[i_f1(i, j + 1, 2, 18, 0)] +
                    pdf[i_f1(i, j - 1, 2, 17, 0)]))) * relaxation_d;

        tnx = prc(0.5) * (
            pdf[i_f1(i - 1, j, 1, 1, 0)] + pdf[i_f1(i - 1, j - 1, 1, 7, 0)] + pdf[i_f1(i - 1, j + 1, 1, 9, 0)] - (
                pdf[i_f1(i + 1, j, 1, 2, 0)] + pdf[i_f1(i + 1, j - 1, 1, 8, 0)] + pdf[i_f1(i + 1, j + 1, 1, 10, 0)]));
        tny = prc(0.5) * (
            pdf[i_f1(i, j - 1, 1, 3, 0)] + pdf[i_f1(i - 1, j - 1, 1, 7, 0)] + pdf[i_f1(i + 1, j - 1, 1, 8, 0)] - (
                pdf[i_f1(i, j + 1, 1, 4, 0)] + pdf[i_f1(i + 1, j + 1, 1, 10, 0)] + pdf[i_f1(i - 1, j + 1, 1, 9, 0)]));

        pdf[i_f1(i, j, 0, 5, 0)] = (pdf[i_f1(i, j, 2, 6, 0)] + prc(0.333333333333333333) * tmp1) * (1 - wall_indicator) + pdf[i_f1(i, j, 0, 5, 0)] * wall_indicator;
        pdf[i_f1(i - 1, j, 0, 11, 0)] = (pdf[i_f1(i + 1, j, 2, 14, 0)] + prc(0.166666666666666667) * tmp1 - tnx) * (1 - wall_indicator) + pdf[i_f1(i - 1, j, 0, 11, 0)] * wall_indicator;
        pdf[i_f1(i + 1, j, 0, 12, 0)] = (pdf[i_f1(i - 1, j, 2, 13, 0)] + prc(0.166666666666666667) * tmp1 + tnx) * (1 - wall_indicator) + pdf[i_f1(i + 1, j, 0, 12, 0)] * wall_indicator;
        pdf[i_f1(i, j - 1, 0, 15, 0)] = (pdf[i_f1(i, j + 1, 2, 18, 0)] + prc(0.166666666666666667) * tmp1 - tny) * (1 - wall_indicator) + pdf[i_f1(i, j - 1, 0, 15, 0)] * wall_indicator;
        pdf[i_f1(i, j + 1, 0, 16, 0)] = (pdf[i_f1(i, j - 1, 2, 17, 0)] + prc(0.166666666666666667) * tmp1 + tny) * (1 - wall_indicator) + pdf[i_f1(i, j + 1, 0, 16, 0)] * wall_indicator;

        tmp2 = (tmpRho2 -
            (pdf[i_f1(i, j, 1, 0, 1)] +
                pdf[i_f1(i - 1, j, 1, 1, 1)] +
                pdf[i_f1(i + 1, j, 1, 2, 1)] +
                pdf[i_f1(i, j - 1, 1, 3, 1)] +
                pdf[i_f1(i, j + 1, 1, 4, 1)] +
                pdf[i_f1(i - 1, j - 1, 1, 7, 1)] +
                pdf[i_f1(i + 1, j - 1, 1, 8, 1)] +
                pdf[i_f1(i - 1, j + 1, 1, 9, 1)] +
                pdf[i_f1(i + 1, j + 1, 1, 10, 1)] + prc(2.) * (
                    pdf[i_f1(i, j, 2, 6, 1)] +
                    pdf[i_f1(i + 1, j, 2, 14, 1)] +
                    pdf[i_f1(i - 1, j, 2, 13, 1)] +
                    pdf[i_f1(i, j + 1, 2, 18, 1)] +
                    pdf[i_f1(i, j - 1, 2, 17, 1)]))) * relaxation_d;

        tnx = prc(0.5) * (
            pdf[i_f1(i - 1, j, 1, 1, 1)] + pdf[i_f1(i - 1, j - 1, 1, 7, 1)] + pdf[i_f1(i - 1, j + 1, 1, 9, 1)] - (
                pdf[i_f1(i + 1, j, 1, 2, 1)] + pdf[i_f1(i + 1, j - 1, 1, 8, 1)] + pdf[i_f1(i + 1, j + 1, 1, 10, 1)]));
        tny = prc(0.5) * (
            pdf[i_f1(i, j - 1, 1, 3, 1)] + pdf[i_f1(i - 1, j - 1, 1, 7, 1)] + pdf[i_f1(i + 1, j - 1, 1, 8, 1)] - (
                pdf[i_f1(i, j + 1, 1, 4, 1)] + pdf[i_f1(i + 1, j + 1, 1, 10, 1)] + pdf[i_f1(i - 1, j + 1, 1, 9, 1)]));

        pdf[i_f1(i, j, 0, 5, 1)] = (pdf[i_f1(i, j, 2, 6, 1)] + prc(0.333333333333333333) * tmp2) * (1 - wall_indicator) + pdf[i_f1(i, j, 0, 5, 1)] * wall_indicator;
        pdf[i_f1(i - 1, j, 0, 11, 1)] = (pdf[i_f1(i + 1, j, 2, 14, 1)] + prc(0.166666666666666667) * tmp2 - tnx) * (1 - wall_indicator) + pdf[i_f1(i - 1, j, 0, 11, 1)] * wall_indicator;
        pdf[i_f1(i + 1, j, 0, 12, 1)] = (pdf[i_f1(i - 1, j, 2, 13, 1)] + prc(0.166666666666666667) * tmp2 + tnx) * (1 - wall_indicator) + pdf[i_f1(i + 1, j, 0, 12, 1)] * wall_indicator;
        pdf[i_f1(i, j - 1, 0, 15, 1)] = (pdf[i_f1(i, j + 1, 2, 18, 1)] + prc(0.166666666666666667) * tmp2 - tny) * (1 - wall_indicator) + pdf[i_f1(i, j - 1, 0, 15, 1)] * wall_indicator;
        pdf[i_f1(i, j + 1, 0, 16, 1)] = (pdf[i_f1(i, j - 1, 2, 17, 1)] + prc(0.166666666666666667) * tmp2 + tny) * (1 - wall_indicator) + pdf[i_f1(i, j + 1, 0, 16, 1)] * wall_indicator;
    }
}

//************************** after odd step kernel *****************************************
__global__ void inlet_Zou_He_pressure_BC_after_odd_GPU(T_P rho_in, int* walls, T_P* phi, T_P* pdf) {

    int i = threadIdx.x + blockIdx.x * blockDim.x + 1;
    int j = threadIdx.y + blockIdx.y * blockDim.y + 1;
    if (i <= nxGlobal_d && j <= nyGlobal_d) {
        int wall_indicator;

        T_P tmp1, tmp2, tnx, tny, tmpRho1, tmpRho2;

        wall_indicator = walls[i_s2(i, j, 1)];

        phi[i_s4(i, j, 0)] = phi_inlet_d * (1 - wall_indicator) + phi[i_s4(i, j, 0)] * wall_indicator;
        phi[i_s4(i, j, -1)] = phi[i_s4(i, j, 0)];
        phi[i_s4(i, j, -2)] = phi[i_s4(i, j, 0)];
        phi[i_s4(i, j, -3)] = phi[i_s4(i, j, 0)];   //overlap_phi = 4

        // inlet pressure BC    k = 1  ~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~
        tmpRho2 = rho_in;
        tmpRho1 = rho_in * sa_inject_d;              // fluid 1 injection
        tmpRho2 = tmpRho2 - tmpRho1;                   // fluid 2 injection

        tmp1 = (tmpRho1 -
            (pdf[i_f1(i, j, 1, 0, 0)] +
                pdf[i_f1(i, j, 1, 2, 0)] +
                pdf[i_f1(i, j, 1, 1, 0)] +
                pdf[i_f1(i, j, 1, 4, 0)] +
                pdf[i_f1(i, j, 1, 3, 0)] +
                pdf[i_f1(i, j, 1, 8, 0)] +
                pdf[i_f1(i, j, 1, 7, 0)] +
                pdf[i_f1(i, j, 1, 10, 0)] +
                pdf[i_f1(i, j, 1, 9, 0)] + prc(2.) * (
                    pdf[i_f1(i, j, 1, 5, 0)] +
                    pdf[i_f1(i, j, 1, 11, 0)] +
                    pdf[i_f1(i, j, 1, 12, 0)] +
                    pdf[i_f1(i, j, 1, 15, 0)] +
                    pdf[i_f1(i, j, 1, 16, 0)]))) * relaxation_d;


        tnx = prc(0.5) * (pdf[i_f1(i, j, 1, 2, 0)] + pdf[i_f1(i, j, 1, 8, 0)] + pdf[i_f1(i, j, 1, 10, 0)] - (pdf[i_f1(i, j, 1, 1, 0)] + pdf[i_f1(i, j, 1, 7, 0)] + pdf[i_f1(i, j, 1, 9, 0)]));
        tny = prc(0.5) * (pdf[i_f1(i, j, 1, 4, 0)] + pdf[i_f1(i, j, 1, 9, 0)] + pdf[i_f1(i, j, 1, 10, 0)] - (pdf[i_f1(i, j, 1, 3, 0)] + pdf[i_f1(i, j, 1, 8, 0)] + pdf[i_f1(i, j, 1, 7, 0)]));

        pdf[i_f1(i, j, 1, 6, 0)] = (pdf[i_f1(i, j, 1, 5, 0)] + prc(0.333333333333333333) * tmp1) * (1 - wall_indicator) + pdf[i_f1(i, j, 1, 6, 0)] * wall_indicator;
        pdf[i_f1(i, j, 1, 13, 0)] = (pdf[i_f1(i, j, 1, 12, 0)] + prc(0.166666666666666667) * tmp1 + tnx) * (1 - wall_indicator) + pdf[i_f1(i, j, 1, 13, 0)] * wall_indicator;
        pdf[i_f1(i, j, 1, 14, 0)] = (pdf[i_f1(i, j, 1, 11, 0)] + prc(0.166666666666666667) * tmp1 - tnx) * (1 - wall_indicator) + pdf[i_f1(i, j, 1, 14, 0)] * wall_indicator;
        pdf[i_f1(i, j, 1, 17, 0)] = (pdf[i_f1(i, j, 1, 16, 0)] + prc(0.166666666666666667) * tmp1 + tny) * (1 - wall_indicator) + pdf[i_f1(i, j, 1, 17, 0)] * wall_indicator;
        pdf[i_f1(i, j, 1, 18, 0)] = (pdf[i_f1(i, j, 1, 15, 0)] + prc(0.166666666666666667) * tmp1 - tny) * (1 - wall_indicator) + pdf[i_f1(i, j, 1, 18, 0)] * wall_indicator;

        tmp2 = (tmpRho2 - (
            pdf[i_f1(i, j, 1, 0, 1)] +
            pdf[i_f1(i, j, 1, 2, 1)] +
            pdf[i_f1(i, j, 1, 1, 1)] +
            pdf[i_f1(i, j, 1, 4, 1)] +
            pdf[i_f1(i, j, 1, 3, 1)] +
            pdf[i_f1(i, j, 1, 8, 1)] +
            pdf[i_f1(i, j, 1, 7, 1)] +
            pdf[i_f1(i, j, 1, 10, 1)] +
            pdf[i_f1(i, j, 1, 9, 1)] + prc(2.) * (
                pdf[i_f1(i, j, 1, 5, 1)] +
                pdf[i_f1(i, j, 1, 11, 1)] +
                pdf[i_f1(i, j, 1, 12, 1)] +
                pdf[i_f1(i, j, 1, 15, 1)] +
                pdf[i_f1(i, j, 1, 16, 1)]))) * relaxation_d;

        tnx = prc(0.5) * (pdf[i_f1(i, j, 1, 2, 1)] + pdf[i_f1(i, j, 1, 8, 1)] + pdf[i_f1(i, j, 1, 10, 1)] - (pdf[i_f1(i, j, 1, 1, 1)] + pdf[i_f1(i, j, 1, 7, 1)] + pdf[i_f1(i, j, 1, 9, 1)]));
        tny = prc(0.5) * (pdf[i_f1(i, j, 1, 4, 1)] + pdf[i_f1(i, j, 1, 9, 1)] + pdf[i_f1(i, j, 1, 10, 1)] - (pdf[i_f1(i, j, 1, 3, 1)] + pdf[i_f1(i, j, 1, 8, 1)] + pdf[i_f1(i, j, 1, 7, 1)]));

        pdf[i_f1(i, j, 1, 6, 1)] = (pdf[i_f1(i, j, 1, 5, 1)] + prc(0.333333333333333333) * tmp2) * (1 - wall_indicator) + pdf[i_f1(i, j, 1, 6, 1)] * wall_indicator;
        pdf[i_f1(i, j, 1, 13, 1)] = (pdf[i_f1(i, j, 1, 12, 1)] + prc(0.166666666666666667) * tmp2 + tnx) * (1 - wall_indicator) + pdf[i_f1(i, j, 1, 13, 1)] * wall_indicator;
        pdf[i_f1(i, j, 1, 14, 1)] = (pdf[i_f1(i, j, 1, 11, 1)] + prc(0.166666666666666667) * tmp2 - tnx) * (1 - wall_indicator) + pdf[i_f1(i, j, 1, 14, 1)] * wall_indicator;
        pdf[i_f1(i, j, 1, 17, 1)] = (pdf[i_f1(i, j, 1, 16, 1)] + prc(0.166666666666666667) * tmp2 + tny) * (1 - wall_indicator) + pdf[i_f1(i, j, 1, 17, 1)] * wall_indicator;
        pdf[i_f1(i, j, 1, 18, 1)] = (pdf[i_f1(i, j, 1, 15, 1)] + prc(0.166666666666666667) * tmp2 - tny) * (1 - wall_indicator) + pdf[i_f1(i, j, 1, 18, 1)] * wall_indicator;
    }
}

#pragma endregion (Inlet Boundary Conditions)

#pragma region (Outlet Boundary Conditions)
__global__ void outlet_convective_BC_before_odd_GPU(int* walls, T_P* phi, T_P* phi_convec_bc, T_P* pdf, T_P* f_convec_bc, T_P* g_convec_bc) {

    int i = threadIdx.x + blockIdx.x * blockDim.x + 1;
    int j = threadIdx.y + blockIdx.y * blockDim.y + 1;
    if (i <= nxGlobal_d && j <= nyGlobal_d) {
        T_P temp, u_convec;
        int wall_indicator;

        u_convec = uin_avg_d;
        temp = prc(1.) / (prc(1.) + u_convec);

        wall_indicator = walls[i_s2(i, j, nzGlobal_d)];

        phi[i_s4(i, j, nzGlobal_d + 1)] = ((phi_convec_bc[i_s1(i, j, 0)] + u_convec * phi[i_s4(i, j, nzGlobal_d)]) * temp) * (1 - wall_indicator)
            + phi[i_s4(i, j, nzGlobal_d + 1)] * wall_indicator;
        phi_convec_bc[i_s1(i, j, 0)] = phi[i_s4(i, j, nzGlobal_d + 1)];   //store PDF for next step
        phi[i_s4(i, j, nzGlobal_d + 2)] = phi[i_s4(i, j, nzGlobal_d + 1)];
        phi[i_s4(i, j, nzGlobal_d + 3)] = phi[i_s4(i, j, nzGlobal_d + 1)];
        phi[i_s4(i, j, nzGlobal_d + 4)] = phi[i_s4(i, j, nzGlobal_d + 1)];   //overlap_phi = 4

        //if outlet convective BC
        pdf[i_f1(i, j, nzGlobal_d + 1, 6, 0)] = ((f_convec_bc[icnv_f1(i, j, 6)] + u_convec * pdf[i_f1(i, j, nzGlobal_d + 1 - 1, 6, 0)]) * temp) * (1 - wall_indicator)
            + pdf[i_f1(i, j, nzGlobal_d + 1, 6, 0)] * wall_indicator;
        pdf[i_f1(i - 1, j, nzGlobal_d + 1, 13, 0)] = ((f_convec_bc[icnv_f1(i, j, 13)] + u_convec * pdf[i_f1(i - 1, j, nzGlobal_d + 1 - 1, 13, 0)]) * temp) * (1 - wall_indicator)
            + pdf[i_f1(i - 1, j, nzGlobal_d + 1, 13, 0)] * wall_indicator;
        pdf[i_f1(i + 1, j, nzGlobal_d + 1, 14, 0)] = ((f_convec_bc[icnv_f1(i, j, 14)] + u_convec * pdf[i_f1(i + 1, j, nzGlobal_d + 1 - 1, 14, 0)]) * temp) * (1 - wall_indicator)
            + pdf[i_f1(i + 1, j, nzGlobal_d + 1, 14, 0)] * wall_indicator;
        pdf[i_f1(i, j - 1, nzGlobal_d + 1, 17, 0)] = ((f_convec_bc[icnv_f1(i, j, 17)] + u_convec * pdf[i_f1(i, j - 1, nzGlobal_d + 1 - 1, 17, 0)]) * temp) * (1 - wall_indicator)
            + pdf[i_f1(i, j - 1, nzGlobal_d + 1, 17, 0)] * wall_indicator;
        pdf[i_f1(i, j + 1, nzGlobal_d + 1, 18, 0)] = ((f_convec_bc[icnv_f1(i, j, 18)] + u_convec * pdf[i_f1(i, j + 1, nzGlobal_d + 1 - 1, 18, 0)]) * temp)
            * (1 - wall_indicator) + pdf[i_f1(i, j + 1, nzGlobal_d + 1, 18, 0)] * wall_indicator;

        pdf[i_f1(i, j, nzGlobal_d + 1, 6, 1)] = ((f_convec_bc[icnv_f1(i, j, 6)] + u_convec * pdf[i_f1(i, j, nzGlobal_d + 1 - 1, 6, 1)]) * temp) * (1 - wall_indicator)
            + pdf[i_f1(i, j, nzGlobal_d + 1, 6, 1)] * wall_indicator;
        pdf[i_f1(i - 1, j, nzGlobal_d + 1, 13, 1)] = ((f_convec_bc[icnv_f1(i, j, 13)] + u_convec * pdf[i_f1(i - 1, j, nzGlobal_d + 1 - 1, 13, 1)]) * temp) * (1 - wall_indicator)
            + pdf[i_f1(i - 1, j, nzGlobal_d + 1, 13, 1)] * wall_indicator;
        pdf[i_f1(i + 1, j, nzGlobal_d + 1, 14, 1)] = ((f_convec_bc[icnv_f1(i, j, 14)] + u_convec * pdf[i_f1(i + 1, j, nzGlobal_d + 1 - 1, 14, 1)]) * temp) * (1 - wall_indicator)
            + pdf[i_f1(i + 1, j, nzGlobal_d + 1, 14, 1)] * wall_indicator;
        pdf[i_f1(i, j - 1, nzGlobal_d + 1, 17, 1)] = ((f_convec_bc[icnv_f1(i, j, 17)] + u_convec * pdf[i_f1(i, j - 1, nzGlobal_d + 1 - 1, 17, 1)]) * temp) * (1 - wall_indicator)
            + pdf[i_f1(i, j - 1, nzGlobal_d + 1, 17, 1)] * wall_indicator;
        pdf[i_f1(i, j + 1, nzGlobal_d + 1, 18, 1)] = ((f_convec_bc[icnv_f1(i, j, 18)] + u_convec * pdf[i_f1(i, j + 1, nzGlobal_d + 1 - 1, 18, 1)]) * temp) * (1 - wall_indicator)
            + pdf[i_f1(i, j + 1, nzGlobal_d + 1, 18, 1)] * wall_indicator;

        f_convec_bc[icnv_f1(i, j, 6)] = pdf[i_f1(i, j, nzGlobal_d + 1, 6, 0)];
        f_convec_bc[icnv_f1(i, j, 13)] = pdf[i_f1(i - 1, j, nzGlobal_d + 1, 13, 0)];
        f_convec_bc[icnv_f1(i, j, 14)] = pdf[i_f1(i + 1, j, nzGlobal_d + 1, 14, 0)];
        f_convec_bc[icnv_f1(i, j, 17)] = pdf[i_f1(i, j - 1, nzGlobal_d + 1, 17, 0)];
        f_convec_bc[icnv_f1(i, j, 18)] = pdf[i_f1(i, j + 1, nzGlobal_d + 1, 18, 0)];

        g_convec_bc[icnv_f1(i, j, 6)] = pdf[i_f1(i, j, nzGlobal_d + 1, 6, 1)];
        g_convec_bc[icnv_f1(i, j, 13)] = pdf[i_f1(i - 1, j, nzGlobal_d + 1, 13, 1)];
        g_convec_bc[icnv_f1(i, j, 14)] = pdf[i_f1(i + 1, j, nzGlobal_d + 1, 14, 1)];
        g_convec_bc[icnv_f1(i, j, 17)] = pdf[i_f1(i, j - 1, nzGlobal_d + 1, 17, 1)];
        g_convec_bc[icnv_f1(i, j, 18)] = pdf[i_f1(i, j + 1, nzGlobal_d + 1, 18, 1)];
    }
}

__global__ void outlet_convective_BC_after_odd_GPU(int* walls, T_P* phi, T_P* phi_convec_bc, T_P* pdf, T_P* f_convec_bc, T_P* g_convec_bc) {
    int i = threadIdx.x + blockIdx.x * blockDim.x + 1;
    int j = threadIdx.y + blockIdx.y * blockDim.y + 1;
    if (i <= nxGlobal_d && j <= nyGlobal_d) {
        T_P temp, u_convec;
        int wall_indicator;

        u_convec = uin_avg_d;
        temp = prc(1.) / (prc(1.) + u_convec);

        wall_indicator = walls[i_s2(i, j, nzGlobal_d)];

        phi[i_s4(i, j, nzGlobal_d + 1)] = ((phi_convec_bc[i_s1(i, j, 0)] + u_convec * phi[i_s4(i, j, nzGlobal_d)]) * temp) * (1 - wall_indicator)
            + phi[i_s4(i, j, nzGlobal_d + 1)] * wall_indicator;
        phi_convec_bc[i_s1(i, j, 0)] = phi[i_s4(i, j, nzGlobal_d + 1)];   //store PDF for next step
        phi[i_s4(i, j, nzGlobal_d + 2)] = phi[i_s4(i, j, nzGlobal_d + 1)];
        phi[i_s4(i, j, nzGlobal_d + 3)] = phi[i_s4(i, j, nzGlobal_d + 1)];
        phi[i_s4(i, j, nzGlobal_d + 4)] = phi[i_s4(i, j, nzGlobal_d + 1)];   //overlap_phi = 4
        // outlet convective bc

        pdf[i_f1(i, j, nzGlobal_d, 5, 0)] = ((f_convec_bc[icnv_f1(i, j, 6)] + u_convec * pdf[i_f1(i, j, nzGlobal_d - 1, 5, 0)]) * temp) * (1 - wall_indicator)
            + pdf[i_f1(i, j, nzGlobal_d, 5, 0)] * wall_indicator;
        pdf[i_f1(i, j, nzGlobal_d, 11, 0)] = ((f_convec_bc[icnv_f1(i, j, 14)] + u_convec * pdf[i_f1(i, j, nzGlobal_d - 1, 11, 0)]) * temp) * (1 - wall_indicator)
            + pdf[i_f1(i, j, nzGlobal_d, 11, 0)] * wall_indicator;
        pdf[i_f1(i, j, nzGlobal_d, 12, 0)] = ((f_convec_bc[icnv_f1(i, j, 13)] + u_convec * pdf[i_f1(i, j, nzGlobal_d - 1, 12, 0)]) * temp) * (1 - wall_indicator)
            + pdf[i_f1(i, j, nzGlobal_d, 12, 0)] * wall_indicator;
        pdf[i_f1(i, j, nzGlobal_d, 15, 0)] = ((f_convec_bc[icnv_f1(i, j, 18)] + u_convec * pdf[i_f1(i, j, nzGlobal_d - 1, 15, 0)]) * temp) * (1 - wall_indicator)
            + pdf[i_f1(i, j, nzGlobal_d, 15, 0)] * wall_indicator;
        pdf[i_f1(i, j, nzGlobal_d, 16, 0)] = ((f_convec_bc[icnv_f1(i, j, 17)] + u_convec * pdf[i_f1(i, j, nzGlobal_d - 1, 16, 0)]) * temp) * (1 - wall_indicator)
            + pdf[i_f1(i, j, nzGlobal_d, 16, 0)] * wall_indicator;

        pdf[i_f1(i, j, nzGlobal_d, 5, 1)] = ((f_convec_bc[icnv_f1(i, j, 6)] + u_convec * pdf[i_f1(i, j, nzGlobal_d - 1, 5, 1)]) * temp) * (1 - wall_indicator)
            + pdf[i_f1(i, j, nzGlobal_d, 5, 1)] * wall_indicator;
        pdf[i_f1(i, j, nzGlobal_d, 11, 1)] = ((f_convec_bc[icnv_f1(i, j, 14)] + u_convec * pdf[i_f1(i, j, nzGlobal_d - 1, 11, 1)]) * temp) * (1 - wall_indicator)
            + pdf[i_f1(i, j, nzGlobal_d, 11, 1)] * wall_indicator;
        pdf[i_f1(i, j, nzGlobal_d, 12, 1)] = ((f_convec_bc[icnv_f1(i, j, 13)] + u_convec * pdf[i_f1(i, j, nzGlobal_d - 1, 12, 1)]) * temp) * (1 - wall_indicator)
            + pdf[i_f1(i, j, nzGlobal_d, 12, 1)] * wall_indicator;
        pdf[i_f1(i, j, nzGlobal_d, 15, 1)] = ((f_convec_bc[icnv_f1(i, j, 18)] + u_convec * pdf[i_f1(i, j, nzGlobal_d - 1, 15, 1)]) * temp) * (1 - wall_indicator)
            + pdf[i_f1(i, j, nzGlobal_d, 15, 1)] * wall_indicator;
        pdf[i_f1(i, j, nzGlobal_d, 16, 1)] = ((f_convec_bc[icnv_f1(i, j, 17)] + u_convec * pdf[i_f1(i, j, nzGlobal_d - 1, 16, 1)]) * temp) * (1 - wall_indicator)
            + pdf[i_f1(i, j, nzGlobal_d, 16, 1)] * wall_indicator;

        f_convec_bc[icnv_f1(i, j, 6)] = pdf[i_f1(i, j, nzGlobal_d, 5, 0)];
        f_convec_bc[icnv_f1(i, j, 14)] = pdf[i_f1(i, j, nzGlobal_d, 11, 0)];
        f_convec_bc[icnv_f1(i, j, 13)] = pdf[i_f1(i, j, nzGlobal_d, 12, 0)];
        f_convec_bc[icnv_f1(i, j, 18)] = pdf[i_f1(i, j, nzGlobal_d, 15, 0)];
        f_convec_bc[icnv_f1(i, j, 17)] = pdf[i_f1(i, j, nzGlobal_d, 16, 0)];

        g_convec_bc[icnv_f1(i, j, 6)] = pdf[i_f1(i, j, nzGlobal_d, 5, 1)];
        g_convec_bc[icnv_f1(i, j, 14)] = pdf[i_f1(i, j, nzGlobal_d, 11, 1)];
        g_convec_bc[icnv_f1(i, j, 13)] = pdf[i_f1(i, j, nzGlobal_d, 12, 1)];
        g_convec_bc[icnv_f1(i, j, 18)] = pdf[i_f1(i, j, nzGlobal_d, 15, 1)];
        g_convec_bc[icnv_f1(i, j, 17)] = pdf[i_f1(i, j, nzGlobal_d, 16, 1)];
    }
}

//=============================================================================================
//----------------------Zou - He type pressure open outlet boundary conditions----------------
//=============================================================================================
//**************************before odd step kernel * ****************************************
__global__ void outlet_Zou_He_pressure_BC_before_odd_GPU(T_P rho_out, int* walls,T_P* phi, T_P* pdf)
    {

    int i = threadIdx.x + blockIdx.x * blockDim.x + 1;
    int j = threadIdx.y + blockIdx.y * blockDim.y + 1;
    if (i <= nxGlobal_d && j <= nyGlobal_d) {
        int wall_indicator;

        T_P tmp1, tmp2, tnx, tny;

        wall_indicator = walls[i_s2(i, j, nzGlobal_d)];
        phi[i_s4(i, j, nzGlobal_d + 1)] = phi[i_s4(i, j, nzGlobal_d)];
        phi[i_s4(i, j, nzGlobal_d + 2)] = phi[i_s4(i, j, nzGlobal_d)];
        phi[i_s4(i, j, nzGlobal_d + 3)] = phi[i_s4(i, j, nzGlobal_d)];
        phi[i_s4(i, j, nzGlobal_d + 4)] = phi[i_s4(i, j, nzGlobal_d)];   // overlap_phi = 4

        // outlet pressure BC    k=1a
        tmp1 = (
            pdf[i_f1(i, j, nzGlobal_d, 0, 0)] +
            pdf[i_f1(i - 1, j, nzGlobal_d, 1, 0)] +
            pdf[i_f1(i + 1, j, nzGlobal_d, 2, 0)] +
            pdf[i_f1(i, j - 1, nzGlobal_d, 3, 0)] +
            pdf[i_f1(i, j + 1, nzGlobal_d, 4, 0)] +
            pdf[i_f1(i - 1, j - 1, nzGlobal_d, 7, 0)] +
            pdf[i_f1(i + 1, j - 1, nzGlobal_d, 8, 0)] +
            pdf[i_f1(i - 1, j + 1, nzGlobal_d, 9, 0)] +
            pdf[i_f1(i + 1, j + 1, nzGlobal_d, 10, 0)] + prc(2.) * (
                pdf[i_f1(i, j, nzGlobal_d - 1, 5, 0)] +
                pdf[i_f1(i - 1, j, nzGlobal_d - 1, 11, 0)] +
                pdf[i_f1(i + 1, j, nzGlobal_d - 1, 12, 0)] +
                pdf[i_f1(i, j - 1, nzGlobal_d - 1, 15, 0)] +
                pdf[i_f1(i, j + 1, nzGlobal_d - 1, 16, 0)]) +
            pdf[i_f1(i, j, nzGlobal_d, 0, 1)] +
            pdf[i_f1(i - 1, j, nzGlobal_d, 1, 1)] +
            pdf[i_f1(i + 1, j, nzGlobal_d, 2, 1)] +
            pdf[i_f1(i, j - 1, nzGlobal_d, 3, 1)] +
            pdf[i_f1(i, j + 1, nzGlobal_d, 4, 1)] +
            pdf[i_f1(i - 1, j - 1, nzGlobal_d, 7, 1)] +
            pdf[i_f1(i + 1, j - 1, nzGlobal_d, 8, 1)] +
            pdf[i_f1(i - 1, j + 1, nzGlobal_d, 9, 1)] +
            pdf[i_f1(i + 1, j + 1, nzGlobal_d, 10, 1)] + prc(2.) * (
                pdf[i_f1(i, j, nzGlobal_d - 1, 5, 1)] +
                pdf[i_f1(i - 1, j, nzGlobal_d - 1, 11, 1)] +
                pdf[i_f1(i + 1, j, nzGlobal_d - 1, 12, 1)] +
                pdf[i_f1(i, j - 1, nzGlobal_d - 1, 15, 1)] +
                pdf[i_f1(i, j + 1, nzGlobal_d - 1, 16, 1)])) - rho_out;


        tmp2 = tmp1 * prc(0.5) * (prc(1.) - phi[i_s4(i, j, nzGlobal_d)]);    // fluid 2 net flux
        tmp1 = tmp1 - tmp2;                   // fluid 1 net flux

        tnx = prc(0.5) * (
            pdf[i_f1(i - 1, j, nzGlobal_d, 1, 0)] + pdf[i_f1(i - 1, j - 1, nzGlobal_d, 7, 0)] + pdf[i_f1(i - 1, j + 1, nzGlobal_d, 9, 0)] - (
                pdf[i_f1(i + 1, j, nzGlobal_d, 2, 0)] + pdf[i_f1(i + 1, j - 1, nzGlobal_d, 8, 0)] + pdf[i_f1(i + 1, j + 1, nzGlobal_d, 10, 0)]));
        tny = prc(0.5) * (
            pdf[i_f1(i, j - 1, nzGlobal_d, 3, 0)] + pdf[i_f1(i - 1, j - 1, nzGlobal_d, 7, 0)] + pdf[i_f1(i + 1, j - 1, nzGlobal_d, 8, 0)] - (
                pdf[i_f1(i, j + 1, nzGlobal_d, 4, 0)] + pdf[i_f1(i + 1, j + 1, nzGlobal_d, 10, 0)] + pdf[i_f1(i - 1, j + 1, nzGlobal_d, 9, 0)]));

        pdf[i_f1(i, j, nzGlobal_d + 1, 6, 0)] = (pdf[i_f1(i, j, nzGlobal_d - 1, 5, 0)] - prc(0.333333333333333333) * tmp1) * (1 - wall_indicator) +
            pdf[i_f1(i, j, nzGlobal_d + 1, 6, 0)] * wall_indicator;
        pdf[i_f1(i - 1, j, nzGlobal_d + 1, 13, 0)] = (pdf[i_f1(i + 1, j, nzGlobal_d - 1, 12, 0)] - prc(0.166666666666666667) * tmp1 - tnx) * (1 - wall_indicator)
            + pdf[i_f1(i - 1, j, nzGlobal_d + 1, 13, 0)] * wall_indicator;
        pdf[i_f1(i + 1, j, nzGlobal_d + 1, 14, 0)] = (pdf[i_f1(i - 1, j, nzGlobal_d - 1, 11, 0)] - prc(0.166666666666666667) * tmp1 + tnx) * (1 - wall_indicator)
            + pdf[i_f1(i + 1, j, nzGlobal_d + 1, 14, 0)] * wall_indicator;
        pdf[i_f1(i, j - 1, nzGlobal_d + 1, 17, 0)] = (pdf[i_f1(i, j + 1, nzGlobal_d - 1, 16, 0)] - prc(0.166666666666666667) * tmp1 - tny) * (1 - wall_indicator)
            + pdf[i_f1(i, j - 1, nzGlobal_d + 1, 17, 0)] * wall_indicator;
        pdf[i_f1(i, j + 1, nzGlobal_d + 1, 18, 0)] = (pdf[i_f1(i, j - 1, nzGlobal_d - 1, 15, 0)] - prc(0.166666666666666667) * tmp1 + tny) * (1 - wall_indicator)
            + pdf[i_f1(i, j + 1, nzGlobal_d + 1, 18, 0)] * wall_indicator;

        tnx = prc(0.5) * (
            pdf[i_f1(i - 1, j, nzGlobal_d, 1, 1)] + pdf[i_f1(i - 1, j - 1, nzGlobal_d, 7, 1)] + pdf[i_f1(i - 1, j + 1, nzGlobal_d, 9, 1)] - (
                pdf[i_f1(i + 1, j, nzGlobal_d, 2, 1)] + pdf[i_f1(i + 1, j - 1, nzGlobal_d, 8, 1)] + pdf[i_f1(i + 1, j + 1, nzGlobal_d, 10, 1)]));
        tny = prc(0.5) * (
            pdf[i_f1(i, j - 1, nzGlobal_d, 3, 1)] + pdf[i_f1(i - 1, j - 1, nzGlobal_d, 7, 1)] + pdf[i_f1(i + 1, j - 1, nzGlobal_d, 8, 1)] - (
                pdf[i_f1(i, j + 1, nzGlobal_d, 4, 1)] + pdf[i_f1(i + 1, j + 1, nzGlobal_d, 10, 1)] + pdf[i_f1(i - 1, j + 1, nzGlobal_d, 9, 1)]));

        pdf[i_f1(i, j, nzGlobal_d + 1, 6, 1)] = (pdf[i_f1(i, j, nzGlobal_d - 1, 5, 1)] - prc(0.333333333333333333) * tmp2) * (1 - wall_indicator) +
            pdf[i_f1(i, j, nzGlobal_d + 1, 6, 1)] * wall_indicator;
        pdf[i_f1(i - 1, j, nzGlobal_d + 1, 13, 1)] = (pdf[i_f1(i + 1, j, nzGlobal_d - 1, 12, 1)] - prc(0.166666666666666667) * tmp2 - tnx) * (1 - wall_indicator)
            + pdf[i_f1(i - 1, j, nzGlobal_d + 1, 13, 1)] * wall_indicator;
        pdf[i_f1(i + 1, j, nzGlobal_d + 1, 14, 1)] = (pdf[i_f1(i - 1, j, nzGlobal_d - 1, 11, 1)] - prc(0.166666666666666667) * tmp2 + tnx) * (1 - wall_indicator)
            + pdf[i_f1(i + 1, j, nzGlobal_d + 1, 14, 1)] * wall_indicator;
        pdf[i_f1(i, j - 1, nzGlobal_d + 1, 17, 1)] = (pdf[i_f1(i, j + 1, nzGlobal_d - 1, 16, 1)] - prc(0.166666666666666667) * tmp2 - tny) * (1 - wall_indicator)
            + pdf[i_f1(i, j - 1, nzGlobal_d + 1, 17, 1)] * wall_indicator;
        pdf[i_f1(i, j + 1, nzGlobal_d + 1, 18, 1)] = (pdf[i_f1(i, j - 1, nzGlobal_d - 1, 15, 1)] - prc(0.166666666666666667) * tmp2 + tny) * (1 - wall_indicator)
            + pdf[i_f1(i, j + 1, nzGlobal_d + 1, 18, 1)] * wall_indicator;
    }
}

// ************************** after odd step kernel *****************************************
__global__ void outlet_Zou_He_pressure_BC_after_odd_GPU(T_P rho_out, int* walls, T_P* phi, T_P* pdf) {

    int i = threadIdx.x + blockIdx.x * blockDim.x + 1;
    int j = threadIdx.y + blockIdx.y * blockDim.y + 1;
    if (i <= nxGlobal_d && j <= nyGlobal_d) {
        int wall_indicator;

        T_P tmp1, tmp2, tnx, tny;

        wall_indicator = walls[i_s2(i, j, nzGlobal_d)];
        phi[i_s4(i, j, nzGlobal_d + 1)] = phi[i_s4(i, j, nzGlobal_d)];
        phi[i_s4(i, j, nzGlobal_d + 2)] = phi[i_s4(i, j, nzGlobal_d)];
        phi[i_s4(i, j, nzGlobal_d + 3)] = phi[i_s4(i, j, nzGlobal_d)];
        phi[i_s4(i, j, nzGlobal_d + 4)] = phi[i_s4(i, j, nzGlobal_d)];   // overlap_phi = 4

        // inlet pressure BC    k=1
        tmp1 = (
            pdf[i_f1(i, j, nzGlobal_d, 0, 0)] +
            pdf[i_f1(i, j, nzGlobal_d, 2, 0)] +
            pdf[i_f1(i, j, nzGlobal_d, 1, 0)] +
            pdf[i_f1(i, j, nzGlobal_d, 4, 0)] +
            pdf[i_f1(i, j, nzGlobal_d, 3, 0)] +
            pdf[i_f1(i, j, nzGlobal_d, 8, 0)] +
            pdf[i_f1(i, j, nzGlobal_d, 7, 0)] +
            pdf[i_f1(i, j, nzGlobal_d, 10, 0)] +
            pdf[i_f1(i, j, nzGlobal_d, 9, 0)] + prc(2.) * (
                pdf[i_f1(i, j, nzGlobal_d, 6, 0)] +
                pdf[i_f1(i, j, nzGlobal_d, 14, 0)] +
                pdf[i_f1(i, j, nzGlobal_d, 13, 0)] +
                pdf[i_f1(i, j, nzGlobal_d, 18, 0)] +
                pdf[i_f1(i, j, nzGlobal_d, 17, 0)]) +
            pdf[i_f1(i, j, nzGlobal_d, 0, 1)] +
            pdf[i_f1(i, j, nzGlobal_d, 2, 1)] +
            pdf[i_f1(i, j, nzGlobal_d, 1, 1)] +
            pdf[i_f1(i, j, nzGlobal_d, 4, 1)] +
            pdf[i_f1(i, j, nzGlobal_d, 3, 1)] +
            pdf[i_f1(i, j, nzGlobal_d, 8, 1)] +
            pdf[i_f1(i, j, nzGlobal_d, 7, 1)] +
            pdf[i_f1(i, j, nzGlobal_d, 10, 1)] +
            pdf[i_f1(i, j, nzGlobal_d, 9, 1)] + prc(2.) * (
                pdf[i_f1(i, j, nzGlobal_d, 6, 1)] +
                pdf[i_f1(i, j, nzGlobal_d, 14, 1)] +
                pdf[i_f1(i, j, nzGlobal_d, 13, 1)] +
                pdf[i_f1(i, j, nzGlobal_d, 18, 1)] +
                pdf[i_f1(i, j, nzGlobal_d, 17, 1)])) - rho_out;

        tmp2 = tmp1 * prc(0.5) * (prc(1.) - phi[i_s4(i, j, nzGlobal_d)]);    //fluid 2 net flux
        tmp1 = tmp1 - tmp2;                   // fluid 1 net flux

        tnx = prc(0.5) * (pdf[i_f1(i, j, nzGlobal_d, 2, 0)] + pdf[i_f1(i, j, nzGlobal_d, 8, 0)] + pdf[i_f1(i, j, nzGlobal_d, 10, 0)]
            - (pdf[i_f1(i, j, nzGlobal_d, 1, 0)] + pdf[i_f1(i, j, nzGlobal_d, 7, 0)] + pdf[i_f1(i, j, nzGlobal_d, 9, 0)]));
        tny = prc(0.5) * (pdf[i_f1(i, j, nzGlobal_d, 4, 0)] + pdf[i_f1(i, j, nzGlobal_d, 10, 0)] + pdf[i_f1(i, j, nzGlobal_d, 9, 0)]
            - (pdf[i_f1(i, j, nzGlobal_d, 3, 0)] + pdf[i_f1(i, j, nzGlobal_d, 7, 0)] + pdf[i_f1(i, j, nzGlobal_d, 8, 0)]));

        pdf[i_f1(i, j, nzGlobal_d, 5, 0)] = (pdf[i_f1(i, j, nzGlobal_d, 6, 0)] - prc(0.333333333333333333) * tmp1) * (1 - wall_indicator) + pdf[i_f1(i, j, nzGlobal_d, 5, 0)] * wall_indicator;
        pdf[i_f1(i, j, nzGlobal_d, 11, 0)] = (pdf[i_f1(i, j, nzGlobal_d, 14, 0)] - prc(0.166666666666666667) * tmp1 + tnx) * (1 - wall_indicator) + pdf[i_f1(i, j, nzGlobal_d, 11, 0)] * wall_indicator;
        pdf[i_f1(i, j, nzGlobal_d, 12, 0)] = (pdf[i_f1(i, j, nzGlobal_d, 13, 0)] - prc(0.166666666666666667) * tmp1 - tnx) * (1 - wall_indicator) + pdf[i_f1(i, j, nzGlobal_d, 12, 0)] * wall_indicator;
        pdf[i_f1(i, j, nzGlobal_d, 15, 0)] = (pdf[i_f1(i, j, nzGlobal_d, 18, 0)] - prc(0.166666666666666667) * tmp1 + tny) * (1 - wall_indicator) + pdf[i_f1(i, j, nzGlobal_d, 15, 0)] * wall_indicator;
        pdf[i_f1(i, j, nzGlobal_d, 16, 0)] = (pdf[i_f1(i, j, nzGlobal_d, 17, 0)] - prc(0.166666666666666667) * tmp1 - tny) * (1 - wall_indicator) + pdf[i_f1(i, j, nzGlobal_d, 16, 0)] * wall_indicator;

        tnx = prc(0.5) * (pdf[i_f1(i, j, nzGlobal_d, 2, 1)] + pdf[i_f1(i, j, nzGlobal_d, 8, 1)] + pdf[i_f1(i, j, nzGlobal_d, 10, 1)]
            - (pdf[i_f1(i, j, nzGlobal_d, 1, 1)] + pdf[i_f1(i, j, nzGlobal_d, 7, 1)] + pdf[i_f1(i, j, nzGlobal_d, 9, 1)]));
        tny = prc(0.5) * (pdf[i_f1(i, j, nzGlobal_d, 4, 1)] + pdf[i_f1(i, j, nzGlobal_d, 10, 1)] + pdf[i_f1(i, j, nzGlobal_d, 9, 1)]
            - (pdf[i_f1(i, j, nzGlobal_d, 3, 1)] + pdf[i_f1(i, j, nzGlobal_d, 7, 1)] + pdf[i_f1(i, j, nzGlobal_d, 8, 1)]));

        pdf[i_f1(i, j, nzGlobal_d, 5, 1)] = (pdf[i_f1(i, j, nzGlobal_d, 6, 1)] - prc(0.333333333333333333) * tmp2) * (1 - wall_indicator) + pdf[i_f1(i, j, nzGlobal_d, 5, 1)] * wall_indicator;
        pdf[i_f1(i, j, nzGlobal_d, 11, 1)] = (pdf[i_f1(i, j, nzGlobal_d, 14, 1)] - prc(0.166666666666666667) * tmp2 + tnx) * (1 - wall_indicator) + pdf[i_f1(i, j, nzGlobal_d, 11, 1)] * wall_indicator;
        pdf[i_f1(i, j, nzGlobal_d, 12, 1)] = (pdf[i_f1(i, j, nzGlobal_d, 13, 1)] - prc(0.166666666666666667) * tmp2 - tnx) * (1 - wall_indicator) + pdf[i_f1(i, j, nzGlobal_d, 12, 1)] * wall_indicator;
        pdf[i_f1(i, j, nzGlobal_d, 15, 1)] = (pdf[i_f1(i, j, nzGlobal_d, 18, 1)] - prc(0.166666666666666667) * tmp2 + tny) * (1 - wall_indicator) + pdf[i_f1(i, j, nzGlobal_d, 15, 1)] * wall_indicator;
        pdf[i_f1(i, j, nzGlobal_d, 16, 1)] = (pdf[i_f1(i, j, nzGlobal_d, 17, 1)] - prc(0.166666666666666667) * tmp2 - tny) * (1 - wall_indicator) + pdf[i_f1(i, j, nzGlobal_d, 16, 1)] * wall_indicator;
    }
}

#pragma endregion (Outlet Boundary Conditions)

#pragma region (Periodic Boundary Conditions)
__global__ void perioidc_BC_Z_even(T_P* pdf) {

    int i = threadIdx.x + blockIdx.x * blockDim.x + 1;
    int j = threadIdx.y + blockIdx.y * blockDim.y + 1;
    if (i <= nxGlobal_d && j <= nyGlobal_d) {
        //****************************z direction******************************************
        pdf[i_f1(i, j, 1 + nzGlobal_d, 6, 0)] = pdf[i_f1(i, j, 1, 6, 0)];
        pdf[i_f1(i, j, 1 + nzGlobal_d, 14, 0)] = pdf[i_f1(i, j, 1, 14, 0)];
        pdf[i_f1(i, j, 1 + nzGlobal_d, 13, 0)] = pdf[i_f1(i, j, 1, 13, 0)];
        pdf[i_f1(i, j, 1 + nzGlobal_d, 18, 0)] = pdf[i_f1(i, j, 1, 18, 0)];
        pdf[i_f1(i, j, 1 + nzGlobal_d, 17, 0)] = pdf[i_f1(i, j, 1, 17, 0)];

        pdf[i_f1(i, j, 1 - 1, 5, 0)] = pdf[i_f1(i, j, nzGlobal_d, 5, 0)];
        pdf[i_f1(i, j, 1 - 1, 11, 0)] = pdf[i_f1(i, j, nzGlobal_d, 11, 0)];
        pdf[i_f1(i, j, 1 - 1, 12, 0)] = pdf[i_f1(i, j, nzGlobal_d, 12, 0)];
        pdf[i_f1(i, j, 1 - 1, 15, 0)] = pdf[i_f1(i, j, nzGlobal_d, 15, 0)];
        pdf[i_f1(i, j, 1 - 1, 16, 0)] = pdf[i_f1(i, j, nzGlobal_d, 16, 0)];

        pdf[i_f1(i, j, 1 + nzGlobal_d, 6, 1)] = pdf[i_f1(i, j, 1, 6, 1)];
        pdf[i_f1(i, j, 1 + nzGlobal_d, 14, 1)] = pdf[i_f1(i, j, 1, 14, 1)];
        pdf[i_f1(i, j, 1 + nzGlobal_d, 13, 1)] = pdf[i_f1(i, j, 1, 13, 1)];
        pdf[i_f1(i, j, 1 + nzGlobal_d, 18, 1)] = pdf[i_f1(i, j, 1, 18, 1)];
        pdf[i_f1(i, j, 1 + nzGlobal_d, 17, 1)] = pdf[i_f1(i, j, 1, 17, 1)];

        pdf[i_f1(i, j, 1 - 1, 5, 1)] = pdf[i_f1(i, j, nzGlobal_d, 5, 1)];
        pdf[i_f1(i, j, 1 - 1, 11, 1)] = pdf[i_f1(i, j, nzGlobal_d, 11, 1)];
        pdf[i_f1(i, j, 1 - 1, 12, 1)] = pdf[i_f1(i, j, nzGlobal_d, 12, 1)];
        pdf[i_f1(i, j, 1 - 1, 15, 1)] = pdf[i_f1(i, j, nzGlobal_d, 15, 1)];
        pdf[i_f1(i, j, 1 - 1, 16, 1)] = pdf[i_f1(i, j, nzGlobal_d, 16, 1)];
    }
}

__global__ void perioidc_BC_Y_even(T_P* pdf) {

    int i = threadIdx.x + blockIdx.x * blockDim.x + 1;
    int k = threadIdx.z + blockIdx.z * blockDim.z + 1;
    if (i <= nxGlobal_d && k <= nzGlobal_d) {
        //****************************y direction******************************************
        pdf[i_f1(i, 1 + nyGlobal_d, k, 4, 0)] = pdf[i_f1(i, 1, k, 4, 0)];
        pdf[i_f1(i, 1 + nyGlobal_d, k, 10, 0)] = pdf[i_f1(i, 1, k, 10, 0)];
        pdf[i_f1(i, 1 + nyGlobal_d, k, 9, 0)] = pdf[i_f1(i, 1, k, 9, 0)];
        pdf[i_f1(i, 1 + nyGlobal_d, k, 16, 0)] = pdf[i_f1(i, 1, k, 16, 0)];
        pdf[i_f1(i, 1 + nyGlobal_d, k, 18, 0)] = pdf[i_f1(i, 1, k, 18, 0)];

        pdf[i_f1(i, 1 - 1, k, 3, 0)] = pdf[i_f1(i, nyGlobal_d, k, 3, 0)];
        pdf[i_f1(i, 1 - 1, k, 7, 0)] = pdf[i_f1(i, nyGlobal_d, k, 7, 0)];
        pdf[i_f1(i, 1 - 1, k, 8, 0)] = pdf[i_f1(i, nyGlobal_d, k, 8, 0)];
        pdf[i_f1(i, 1 - 1, k, 15, 0)] = pdf[i_f1(i, nyGlobal_d, k, 15, 0)];
        pdf[i_f1(i, 1 - 1, k, 17, 0)] = pdf[i_f1(i, nyGlobal_d, k, 17, 0)];

        pdf[i_f1(i, 1 + nyGlobal_d, k, 4, 1)] = pdf[i_f1(i, 1, k, 4, 1)];
        pdf[i_f1(i, 1 + nyGlobal_d, k, 10, 1)] = pdf[i_f1(i, 1, k, 10, 1)];
        pdf[i_f1(i, 1 + nyGlobal_d, k, 9, 1)] = pdf[i_f1(i, 1, k, 9, 1)];
        pdf[i_f1(i, 1 + nyGlobal_d, k, 16, 1)] = pdf[i_f1(i, 1, k, 16, 1)];
        pdf[i_f1(i, 1 + nyGlobal_d, k, 18, 1)] = pdf[i_f1(i, 1, k, 18, 1)];

        pdf[i_f1(i, 1 - 1, k, 3, 1)] = pdf[i_f1(i, nyGlobal_d, k, 3, 1)];
        pdf[i_f1(i, 1 - 1, k, 7, 1)] = pdf[i_f1(i, nyGlobal_d, k, 7, 1)];
        pdf[i_f1(i, 1 - 1, k, 8, 1)] = pdf[i_f1(i, nyGlobal_d, k, 8, 1)];
        pdf[i_f1(i, 1 - 1, k, 15, 1)] = pdf[i_f1(i, nyGlobal_d, k, 15, 1)];
        pdf[i_f1(i, 1 - 1, k, 17, 1)] = pdf[i_f1(i, nyGlobal_d, k, 17, 1)];
    }
}

__global__ void perioidc_BC_ZY_even(T_P* pdf) {

    int i = threadIdx.x + blockIdx.x * blockDim.x + 1;
    if (i <= nxGlobal_d) {
        // ****************************edges******************************************
        pdf[i_f1(i, nyGlobal_d + 1, nzGlobal_d + 1, 18, 0)] = pdf[i_f1(i, 1, 1, 18, 0)];
        pdf[i_f1(i, nyGlobal_d + 1, 1 - 1, 16, 0)] = pdf[i_f1(i, 1, nzGlobal_d, 16, 0)];
        pdf[i_f1(i, 1 - 1, nzGlobal_d + 1, 17, 0)] = pdf[i_f1(i, nyGlobal_d, 1, 17, 0)];
        pdf[i_f1(i, 1 - 1, 1 - 1, 15, 0)] = pdf[i_f1(i, nyGlobal_d, nzGlobal_d, 15, 0)];

        pdf[i_f1(i, nyGlobal_d + 1, nzGlobal_d + 1, 18, 1)] = pdf[i_f1(i, 1, 1, 18, 1)];
        pdf[i_f1(i, nyGlobal_d + 1, 1 - 1, 16, 1)] = pdf[i_f1(i, 1, nzGlobal_d, 16, 1)];
        pdf[i_f1(i, 1 - 1, nzGlobal_d + 1, 17, 1)] = pdf[i_f1(i, nyGlobal_d, 1, 17, 1)];
        pdf[i_f1(i, 1 - 1, 1 - 1, 15, 1)] = pdf[i_f1(i, nyGlobal_d, nzGlobal_d, 15, 1)];
    }
}

__global__ void perioidc_BC_Z_odd(T_P* pdf) {

    int i = threadIdx.x + blockIdx.x * blockDim.x + 1;
    int j = threadIdx.y + blockIdx.y * blockDim.y + 1;
    if (i <= nxGlobal_d && j <= nyGlobal_d) {
        //****************************z direction******************************************
        pdf[i_f1(i, j, 1, 6, 0)] = pdf[i_f1(i, j, nzGlobal_d + 1, 6, 0)];
        pdf[i_f1(i, j, 1, 14, 0)] = pdf[i_f1(i, j, nzGlobal_d + 1, 14, 0)];
        pdf[i_f1(i, j, 1, 13, 0)] = pdf[i_f1(i, j, nzGlobal_d + 1, 13, 0)];
        pdf[i_f1(i, j, 1, 18, 0)] = pdf[i_f1(i, j, nzGlobal_d + 1, 18, 0)];
        pdf[i_f1(i, j, 1, 17, 0)] = pdf[i_f1(i, j, nzGlobal_d + 1, 17, 0)];

        pdf[i_f1(i, j, nzGlobal_d, 5, 0)] = pdf[i_f1(i, j, 1 - 1, 5, 0)];
        pdf[i_f1(i, j, nzGlobal_d, 11, 0)] = pdf[i_f1(i, j, 1 - 1, 11, 0)];
        pdf[i_f1(i, j, nzGlobal_d, 12, 0)] = pdf[i_f1(i, j, 1 - 1, 12, 0)];
        pdf[i_f1(i, j, nzGlobal_d, 15, 0)] = pdf[i_f1(i, j, 1 - 1, 15, 0)];
        pdf[i_f1(i, j, nzGlobal_d, 16, 0)] = pdf[i_f1(i, j, 1 - 1, 16, 0)];

        pdf[i_f1(i, j, 1, 6, 1)] = pdf[i_f1(i, j, nzGlobal_d + 1, 6, 1)];
        pdf[i_f1(i, j, 1, 14, 1)] = pdf[i_f1(i, j, nzGlobal_d + 1, 14, 1)];
        pdf[i_f1(i, j, 1, 13, 1)] = pdf[i_f1(i, j, nzGlobal_d + 1, 13, 1)];
        pdf[i_f1(i, j, 1, 18, 1)] = pdf[i_f1(i, j, nzGlobal_d + 1, 18, 1)];
        pdf[i_f1(i, j, 1, 17, 1)] = pdf[i_f1(i, j, nzGlobal_d + 1, 17, 1)];

        pdf[i_f1(i, j, nzGlobal_d, 5, 1)] = pdf[i_f1(i, j, 1 - 1, 5, 1)];
        pdf[i_f1(i, j, nzGlobal_d, 11, 1)] = pdf[i_f1(i, j, 1 - 1, 11, 1)];
        pdf[i_f1(i, j, nzGlobal_d, 12, 1)] = pdf[i_f1(i, j, 1 - 1, 12, 1)];
        pdf[i_f1(i, j, nzGlobal_d, 15, 1)] = pdf[i_f1(i, j, 1 - 1, 15, 1)];
        pdf[i_f1(i, j, nzGlobal_d, 16, 1)] = pdf[i_f1(i, j, 1 - 1, 16, 1)];
    }
}

__global__ void perioidc_BC_Y_odd(T_P* pdf) {

    int i = threadIdx.x + blockIdx.x * blockDim.x + 1;
    int k = threadIdx.z + blockIdx.z * blockDim.z + 1;
    if (i <= nxGlobal_d && k <= nzGlobal_d) {
        //****************************y direction******************************************
        pdf[i_f1(i, 1, k, 4, 0)] = pdf[i_f1(i, nyGlobal_d + 1, k, 4, 0)];
        pdf[i_f1(i, 1, k, 10, 0)] = pdf[i_f1(i, nyGlobal_d + 1, k, 10, 0)];
        pdf[i_f1(i, 1, k, 9, 0)] = pdf[i_f1(i, nyGlobal_d + 1, k, 9, 0)];
        pdf[i_f1(i, 1, k, 16, 0)] = pdf[i_f1(i, nyGlobal_d + 1, k, 16, 0)];
        pdf[i_f1(i, 1, k, 18, 0)] = pdf[i_f1(i, nyGlobal_d + 1, k, 18, 0)];

        pdf[i_f1(i, nyGlobal_d, k, 3, 0)] = pdf[i_f1(i, 1 - 1, k, 3, 0)];
        pdf[i_f1(i, nyGlobal_d, k, 7, 0)] = pdf[i_f1(i, 1 - 1, k, 7, 0)];
        pdf[i_f1(i, nyGlobal_d, k, 8, 0)] = pdf[i_f1(i, 1 - 1, k, 8, 0)];
        pdf[i_f1(i, nyGlobal_d, k, 15, 0)] = pdf[i_f1(i, 1 - 1, k, 15, 0)];
        pdf[i_f1(i, nyGlobal_d, k, 17, 0)] = pdf[i_f1(i, 1 - 1, k, 17, 0)];

        pdf[i_f1(i, 1, k, 4, 1)] = pdf[i_f1(i, nyGlobal_d + 1, k, 4, 1)];
        pdf[i_f1(i, 1, k, 10, 1)] = pdf[i_f1(i, nyGlobal_d + 1, k, 10, 1)];
        pdf[i_f1(i, 1, k, 9, 1)] = pdf[i_f1(i, nyGlobal_d + 1, k, 9, 1)];
        pdf[i_f1(i, 1, k, 16, 1)] = pdf[i_f1(i, nyGlobal_d + 1, k, 16, 1)];
        pdf[i_f1(i, 1, k, 18, 1)] = pdf[i_f1(i, nyGlobal_d + 1, k, 18, 1)];

        pdf[i_f1(i, nyGlobal_d, k, 3, 1)] = pdf[i_f1(i, 1 - 1, k, 3, 1)];
        pdf[i_f1(i, nyGlobal_d, k, 7, 1)] = pdf[i_f1(i, 1 - 1, k, 7, 1)];
        pdf[i_f1(i, nyGlobal_d, k, 8, 1)] = pdf[i_f1(i, 1 - 1, k, 8, 1)];
        pdf[i_f1(i, nyGlobal_d, k, 15, 1)] = pdf[i_f1(i, 1 - 1, k, 15, 1)];
        pdf[i_f1(i, nyGlobal_d, k, 17, 1)] = pdf[i_f1(i, 1 - 1, k, 17, 1)];
    }
}

__global__ void perioidc_BC_ZY_odd(T_P* pdf) {

    int i = threadIdx.x + blockIdx.x * blockDim.x + 1;
    if (i <= nxGlobal_d) {
        // ****************************edges******************************************
        pdf[i_f1(i, 1, 1, 18, 0)] = pdf[i_f1(i, nyGlobal_d + 1, nzGlobal_d + 1, 18, 0)];
        pdf[i_f1(i, 1, nzGlobal_d, 16, 0)] = pdf[i_f1(i, nyGlobal_d + 1, 1 - 1, 16, 0)];
        pdf[i_f1(i, nyGlobal_d, 1, 17, 0)] = pdf[i_f1(i, 1 - 1, nzGlobal_d + 1, 17, 0)];
        pdf[i_f1(i, nyGlobal_d, nzGlobal_d, 15, 0)] = pdf[i_f1(i, 1 - 1, 1 - 1, 15, 0)];

        pdf[i_f1(i, 1, 1, 18, 1)] = pdf[i_f1(i, nyGlobal_d + 1, nzGlobal_d + 1, 18, 1)];
        pdf[i_f1(i, 1, nzGlobal_d, 16, 1)] = pdf[i_f1(i, nyGlobal_d + 1, 1 - 1, 16, 1)];
        pdf[i_f1(i, nyGlobal_d, 1, 17, 1)] = pdf[i_f1(i, 1 - 1, nzGlobal_d + 1, 17, 1)];
        pdf[i_f1(i, nyGlobal_d, nzGlobal_d, 15, 1)] = pdf[i_f1(i, 1 - 1, 1 - 1, 15, 1)];
    }
}

__global__ void periodic_phi_Z(int overlap_phi, T_P* phi) {

    int i = threadIdx.x + blockIdx.x * blockDim.x + 1;
    int j = threadIdx.y + blockIdx.y * blockDim.y + 1;
    if (i <= nxGlobal_d && j <= nyGlobal_d) {
        // **************************** faces ******************************************
        int k;
        for (k = 1; k <= overlap_phi; k++) {
            phi[i_s4(i, j, k + nzGlobal_d)] = phi[i_s4(i, j, k)];
            phi[i_s4(i, j, k - overlap_phi)] = phi[i_s4(i, j, nzGlobal_d + k - overlap_phi)];
        }
    }
}
__global__ void periodic_phi_Y(int overlap_phi, T_P* phi) {

    int i = threadIdx.x + blockIdx.x * blockDim.x + 1;
    int k = threadIdx.z + blockIdx.z * blockDim.z + 1;
    if (i <= nxGlobal_d && k <= nzGlobal_d) {
        // **************************** faces ******************************************
        int j;
        for (j = 1; j <= overlap_phi; j++) {
            phi[i_s4(i, j + nyGlobal_d, k)] = phi[i_s4(i, j, k)];
            phi[i_s4(i, j - overlap_phi, k)] = phi[i_s4(i, nyGlobal_d + j - overlap_phi, k)];
        }
    }
}

__global__ void periodic_phi_ZY(int overlap_phi, T_P* phi) {

    int i = threadIdx.x + blockIdx.x * blockDim.x + 1;
    if (i <= nxGlobal_d) {
        // **************************** edges ******************************************
        int k, j;
        for (k = 1; k <= overlap_phi; k++) {
            for (j = 1; j <= overlap_phi; j++) {
                phi[i_s4(i, j - overlap_phi, k - overlap_phi)] = phi[i_s4(i, nyGlobal_d + j - overlap_phi, nzGlobal_d + k - overlap_phi)];
                phi[i_s4(i, j + nyGlobal_d, k - overlap_phi)] = phi[i_s4(i, j, nzGlobal_d + k - overlap_phi)];
                phi[i_s4(i, j + nyGlobal_d, k + nzGlobal_d)] = phi[i_s4(i, j, k)];
                phi[i_s4(i, j - overlap_phi, k + nzGlobal_d)] = phi[i_s4(i, nyGlobal_d + j - overlap_phi, k)];
            }
        }
    }
}

#pragma endregion (Periodic Boundary Conditions)

#pragma region (Other Boundary Conditions)
//===================================================================================================================================================================================
//----------------------place a porous plate in the domain : 0 - no; 1 - block fluid 1; 2 - block fluid 2  ----------------------
//example1: inject fluid 2 (wetting)during imbibition cycle and block fluid 1 from exiting the inlet
//example2 : inject fluid 1 (nonwetting)during drainage cycle and block fluid 1 from exiting the outlet
//===================================================================================================================================================================================
//**************************before odd step kernel * ****************************************
__global__ void porous_plate_BC_before_odd(T_P* pdf) { // before streaming type BC
    int i = threadIdx.x + blockIdx.x * blockDim.x + 1;
    int j = threadIdx.y + blockIdx.y * blockDim.y + 1;

    if (i <= nxGlobal_d && j <= nyGlobal_d) {
        int zmin, zmax, z_porous_plate_local;

        zmin = 1;
        zmax = nzGlobal_d;

        if (Z_porous_plate_d >= zmin && Z_porous_plate_d <= zmax) {
            z_porous_plate_local = Z_porous_plate_d;
            if (porous_plate_cmd_d == 1) { // block fluid 1, default (assuming fluid 1 is the nonwetting phase)

                pdf[i_f1(i, j, z_porous_plate_local, 6, 0)] = pdf[i_f1(i, j, z_porous_plate_local - 1, 5, 0)];
                pdf[i_f1(i - 1, j, z_porous_plate_local, 13, 0)] = pdf[i_f1(i, j, z_porous_plate_local - 1, 12, 0)];
                pdf[i_f1(i + 1, j, z_porous_plate_local, 14, 0)] = pdf[i_f1(i, j, z_porous_plate_local - 1, 11, 0)];
                pdf[i_f1(i, j - 1, z_porous_plate_local, 17, 0)] = pdf[i_f1(i, j, z_porous_plate_local - 1, 16, 0)];
                pdf[i_f1(i, j + 1, z_porous_plate_local, 18, 0)] = pdf[i_f1(i, j, z_porous_plate_local - 1, 15, 0)];

                pdf[i_f1(i, j, z_porous_plate_local, 5, 0)] = pdf[i_f1(i, j, z_porous_plate_local + 1, 6, 0)];
                pdf[i_f1(i + 1, j, z_porous_plate_local, 12, 0)] = pdf[i_f1(i, j, z_porous_plate_local + 1, 13, 0)];
                pdf[i_f1(i - 1, j, z_porous_plate_local, 11, 0)] = pdf[i_f1(i, j, z_porous_plate_local + 1, 14, 0)];
                pdf[i_f1(i, j + 1, z_porous_plate_local, 16, 0)] = pdf[i_f1(i, j, z_porous_plate_local + 1, 17, 0)];
                pdf[i_f1(i, j - 1, z_porous_plate_local, 15, 0)] = pdf[i_f1(i, j, z_porous_plate_local + 1, 18, 0)];

                pdf[i_f1(i, j, z_porous_plate_local, 6, 1)] = pdf[i_f1(i, j, z_porous_plate_local + 1, 6, 1)];
                pdf[i_f1(i, j, z_porous_plate_local, 13, 1)] = pdf[i_f1(i, j, z_porous_plate_local + 1, 13, 1)];
                pdf[i_f1(i, j, z_porous_plate_local, 14, 1)] = pdf[i_f1(i, j, z_porous_plate_local + 1, 14, 1)];
                pdf[i_f1(i, j, z_porous_plate_local, 17, 1)] = pdf[i_f1(i, j, z_porous_plate_local + 1, 17, 1)];
                pdf[i_f1(i, j, z_porous_plate_local, 18, 1)] = pdf[i_f1(i, j, z_porous_plate_local + 1, 18, 1)];

                pdf[i_f1(i, j, z_porous_plate_local, 5, 1)] = pdf[i_f1(i, j, z_porous_plate_local - 1, 5, 1)];
                pdf[i_f1(i, j, z_porous_plate_local, 12, 1)] = pdf[i_f1(i, j, z_porous_plate_local - 1, 12, 1)];
                pdf[i_f1(i, j, z_porous_plate_local, 11, 1)] = pdf[i_f1(i, j, z_porous_plate_local - 1, 11, 1)];
                pdf[i_f1(i, j, z_porous_plate_local, 16, 1)] = pdf[i_f1(i, j, z_porous_plate_local - 1, 16, 1)];
                pdf[i_f1(i, j, z_porous_plate_local, 15, 1)] = pdf[i_f1(i, j, z_porous_plate_local - 1, 15, 1)];

            }
            else if (porous_plate_cmd_d == 2) { // block fluid 2

                pdf[i_f1(i, j, z_porous_plate_local, 6, 1)] = pdf[i_f1(i, j, z_porous_plate_local - 1, 5, 1)];
                pdf[i_f1(i - 1, j, z_porous_plate_local, 13, 1)] = pdf[i_f1(i, j, z_porous_plate_local - 1, 12, 1)];
                pdf[i_f1(i + 1, j, z_porous_plate_local, 14, 1)] = pdf[i_f1(i, j, z_porous_plate_local - 1, 11, 1)];
                pdf[i_f1(i, j - 1, z_porous_plate_local, 17, 1)] = pdf[i_f1(i, j, z_porous_plate_local - 1, 16, 1)];
                pdf[i_f1(i, j + 1, z_porous_plate_local, 18, 1)] = pdf[i_f1(i, j, z_porous_plate_local - 1, 15, 1)];

                pdf[i_f1(i, j, z_porous_plate_local, 5, 1)] = pdf[i_f1(i, j, z_porous_plate_local + 1, 6, 1)];
                pdf[i_f1(i + 1, j, z_porous_plate_local, 12, 1)] = pdf[i_f1(i, j, z_porous_plate_local + 1, 13, 1)];
                pdf[i_f1(i - 1, j, z_porous_plate_local, 11, 1)] = pdf[i_f1(i, j, z_porous_plate_local + 1, 14, 1)];
                pdf[i_f1(i, j + 1, z_porous_plate_local, 16, 1)] = pdf[i_f1(i, j, z_porous_plate_local + 1, 17, 1)];
                pdf[i_f1(i, j - 1, z_porous_plate_local, 15, 1)] = pdf[i_f1(i, j, z_porous_plate_local + 1, 18, 1)];

                pdf[i_f1(i, j, z_porous_plate_local, 6, 0)] = pdf[i_f1(i, j, z_porous_plate_local + 1, 6, 0)];
                pdf[i_f1(i, j, z_porous_plate_local, 13, 0)] = pdf[i_f1(i, j, z_porous_plate_local + 1, 13, 0)];
                pdf[i_f1(i, j, z_porous_plate_local, 14, 0)] = pdf[i_f1(i, j, z_porous_plate_local + 1, 14, 0)];
                pdf[i_f1(i, j, z_porous_plate_local, 17, 0)] = pdf[i_f1(i, j, z_porous_plate_local + 1, 17, 0)];
                pdf[i_f1(i, j, z_porous_plate_local, 18, 0)] = pdf[i_f1(i, j, z_porous_plate_local + 1, 18, 0)];

                pdf[i_f1(i, j, z_porous_plate_local, 5, 0)] = pdf[i_f1(i, j, z_porous_plate_local - 1, 5, 0)];
                pdf[i_f1(i, j, z_porous_plate_local, 12, 0)] = pdf[i_f1(i, j, z_porous_plate_local - 1, 12, 0)];
                pdf[i_f1(i, j, z_porous_plate_local, 11, 0)] = pdf[i_f1(i, j, z_porous_plate_local - 1, 11, 0)];
                pdf[i_f1(i, j, z_porous_plate_local, 16, 0)] = pdf[i_f1(i, j, z_porous_plate_local - 1, 16, 0)];
                pdf[i_f1(i, j, z_porous_plate_local, 15, 0)] = pdf[i_f1(i, j, z_porous_plate_local - 1, 15, 0)];

            } 

        }
    }
}

// ************************** after odd step kernel *****************************************
__global__ void porous_plate_BC_after_odd(T_P* pdf) { // after streaming type BC
    int i = threadIdx.x + blockIdx.x * blockDim.x + 1;
    int j = threadIdx.y + blockIdx.y * blockDim.y + 1;
    if (i <= nxGlobal_d && j <= nyGlobal_d) {
        int zmin, zmax, z_porous_plate_local;

        zmin = 1;
        zmax = nzGlobal_d;

        if (Z_porous_plate_d >= zmin && Z_porous_plate_d <= zmax) {
            z_porous_plate_local = Z_porous_plate_d;
            if (porous_plate_cmd_d == 1) { // block fluid 1, default (assuming fluid 1 is the nonwetting phase)
                pdf[i_f1(i, j, z_porous_plate_local - 1, 5, 0)] = pdf[i_f1(i, j, z_porous_plate_local, 6, 0)];
                pdf[i_f1(i, j, z_porous_plate_local - 1, 12, 0)] = pdf[i_f1(i - 1, j, z_porous_plate_local, 13, 0)];
                pdf[i_f1(i, j, z_porous_plate_local - 1, 11, 0)] = pdf[i_f1(i + 1, j, z_porous_plate_local, 14, 0)];
                pdf[i_f1(i, j, z_porous_plate_local - 1, 16, 0)] = pdf[i_f1(i, j - 1, z_porous_plate_local, 17, 0)];
                pdf[i_f1(i, j, z_porous_plate_local - 1, 15, 0)] = pdf[i_f1(i, j + 1, z_porous_plate_local, 18, 0)];

                pdf[i_f1(i, j, z_porous_plate_local + 1, 6, 0)] = pdf[i_f1(i, j, z_porous_plate_local, 5, 0)];
                pdf[i_f1(i, j, z_porous_plate_local + 1, 13, 0)] = pdf[i_f1(i + 1, j, z_porous_plate_local, 12, 0)];
                pdf[i_f1(i, j, z_porous_plate_local + 1, 14, 0)] = pdf[i_f1(i - 1, j, z_porous_plate_local, 11, 0)];
                pdf[i_f1(i, j, z_porous_plate_local + 1, 17, 0)] = pdf[i_f1(i, j + 1, z_porous_plate_local, 16, 0)];
                pdf[i_f1(i, j, z_porous_plate_local + 1, 18, 0)] = pdf[i_f1(i, j - 1, z_porous_plate_local, 15, 0)];

                pdf[i_f1(i, j, z_porous_plate_local - 1, 5, 1)] = pdf[i_f1(i, j, z_porous_plate_local, 5, 1)];
                pdf[i_f1(i, j, z_porous_plate_local - 1, 12, 1)] = pdf[i_f1(i, j, z_porous_plate_local, 12, 1)];
                pdf[i_f1(i, j, z_porous_plate_local - 1, 11, 1)] = pdf[i_f1(i, j, z_porous_plate_local, 11, 1)];
                pdf[i_f1(i, j, z_porous_plate_local - 1, 16, 1)] = pdf[i_f1(i, j, z_porous_plate_local, 16, 1)];
                pdf[i_f1(i, j, z_porous_plate_local - 1, 15, 1)] = pdf[i_f1(i, j, z_porous_plate_local, 15, 1)];

                pdf[i_f1(i, j, z_porous_plate_local + 1, 6, 1)] = pdf[i_f1(i, j, z_porous_plate_local, 6, 1)];
                pdf[i_f1(i, j, z_porous_plate_local + 1, 13, 1)] = pdf[i_f1(i, j, z_porous_plate_local, 13, 1)];
                pdf[i_f1(i, j, z_porous_plate_local + 1, 14, 1)] = pdf[i_f1(i, j, z_porous_plate_local, 14, 1)];
                pdf[i_f1(i, j, z_porous_plate_local + 1, 17, 1)] = pdf[i_f1(i, j, z_porous_plate_local, 17, 1)];
                pdf[i_f1(i, j, z_porous_plate_local + 1, 18, 1)] = pdf[i_f1(i, j, z_porous_plate_local, 18, 1)];
            }
            else if (porous_plate_cmd_d == 2) { // block fluid 2

                pdf[i_f1(i, j, z_porous_plate_local - 1, 5, 1)] = pdf[i_f1(i, j, z_porous_plate_local, 6, 1)];
                pdf[i_f1(i, j, z_porous_plate_local - 1, 12, 1)] = pdf[i_f1(i - 1, j, z_porous_plate_local, 13, 1)];
                pdf[i_f1(i, j, z_porous_plate_local - 1, 11, 1)] = pdf[i_f1(i + 1, j, z_porous_plate_local, 14, 1)];
                pdf[i_f1(i, j, z_porous_plate_local - 1, 16, 1)] = pdf[i_f1(i, j - 1, z_porous_plate_local, 17, 1)];
                pdf[i_f1(i, j, z_porous_plate_local - 1, 15, 1)] = pdf[i_f1(i, j + 1, z_porous_plate_local, 18, 1)];

                pdf[i_f1(i, j, z_porous_plate_local + 1, 6, 1)] = pdf[i_f1(i, j, z_porous_plate_local, 5, 1)];
                pdf[i_f1(i, j, z_porous_plate_local + 1, 13, 1)] = pdf[i_f1(i + 1, j, z_porous_plate_local, 12, 1)];
                pdf[i_f1(i, j, z_porous_plate_local + 1, 14, 1)] = pdf[i_f1(i - 1, j, z_porous_plate_local, 11, 1)];
                pdf[i_f1(i, j, z_porous_plate_local + 1, 17, 1)] = pdf[i_f1(i, j + 1, z_porous_plate_local, 16, 1)];
                pdf[i_f1(i, j, z_porous_plate_local + 1, 18, 1)] = pdf[i_f1(i, j - 1, z_porous_plate_local, 15, 1)];

                pdf[i_f1(i, j, z_porous_plate_local + 1, 6, 0)] = pdf[i_f1(i, j, z_porous_plate_local, 6, 0)];
                pdf[i_f1(i, j, z_porous_plate_local + 1, 13, 0)] = pdf[i_f1(i, j, z_porous_plate_local, 13, 0)];
                pdf[i_f1(i, j, z_porous_plate_local + 1, 14, 0)] = pdf[i_f1(i, j, z_porous_plate_local, 14, 0)];
                pdf[i_f1(i, j, z_porous_plate_local + 1, 17, 0)] = pdf[i_f1(i, j, z_porous_plate_local, 17, 0)];
                pdf[i_f1(i, j, z_porous_plate_local + 1, 18, 0)] = pdf[i_f1(i, j, z_porous_plate_local, 18, 0)];

                pdf[i_f1(i, j, z_porous_plate_local - 1, 5, 0)] = pdf[i_f1(i, j, z_porous_plate_local, 5, 0)];
                pdf[i_f1(i, j, z_porous_plate_local - 1, 12, 0)] = pdf[i_f1(i, j, z_porous_plate_local, 12, 0)];
                pdf[i_f1(i, j, z_porous_plate_local - 1, 11, 0)] = pdf[i_f1(i, j, z_porous_plate_local, 11, 0)];
                pdf[i_f1(i, j, z_porous_plate_local - 1, 16, 0)] = pdf[i_f1(i, j, z_porous_plate_local, 16, 0)];
                pdf[i_f1(i, j, z_porous_plate_local - 1, 15, 0)] = pdf[i_f1(i, j, z_porous_plate_local, 15, 0)];

            }
        }

    }
}


#pragma endregion (Other Boundary Conditions)

#pragma endregion (Boundary Conditions)


void main_iteration_kernel_GPU() {
    /* define grid structure */
    dim3 block;
    dim3 grid;

    if (ntime % 2 == 0) {
        /* ************************** even step ***************************************** */
        block.x = block_Threads_X;    block.y = block_Threads_Y;    block.z = block_Threads_Z;
        grid.x = (int(nxGlobal) + block.x - 1) / block.x; grid.y = (int(nyGlobal) + block.y - 1) / block.y; grid.z = (int(nzGlobal) + block.z - 1) / block.z;
        kernel_even_color_GPU << <grid, block >> > (1, int(nxGlobal), 1, int(nyGlobal), 1, int(nzGlobal), walls_d, pdf_d, phi_d, cn_x_d, cn_y_d, cn_z_d, curv_d, c_norm_d);
        cudaErrorCheck(hipDeviceSynchronize()); cudaErrorCheck(hipPeekAtLastError());
        /* ************************** even step ***************************************** */
        /* ************************** periodic boundary conditions  ***************************************** */
        if (kper) {
            block.x = block_Threads_X;    block.y = block_Threads_Y;    block.z = 1;
            grid.x = (int(nxGlobal) + block.x - 1) / block.x; grid.y = (int(nyGlobal) + block.y - 1) / block.y; grid.z = 1;
            perioidc_BC_Z_even << <grid, block >> > (pdf_d);
            cudaErrorCheck(hipDeviceSynchronize()); cudaErrorCheck(hipPeekAtLastError());
            periodic_phi_Z << <grid, block >> > (4, phi_d);
            cudaErrorCheck(hipDeviceSynchronize()); cudaErrorCheck(hipPeekAtLastError());
        }
        if (jper) {
            block.x = block_Threads_X;    block.y = 1;    block.z = block_Threads_Z;
            grid.x = (int(nxGlobal) + block.x - 1) / block.x; grid.y = 1; grid.z = (int(nzGlobal) + block.z - 1) / block.z;   
            perioidc_BC_Y_even << <grid, block >> > (pdf_d);
            cudaErrorCheck(hipDeviceSynchronize()); cudaErrorCheck(hipPeekAtLastError());
            periodic_phi_Y << <grid, block >> > (4, phi_d);
            cudaErrorCheck(hipDeviceSynchronize()); cudaErrorCheck(hipPeekAtLastError());

        }
        if (jper && kper) {
            block.x = block_Threads_X; block.y = 1; block.z = 1;
            grid.x = (int(nxGlobal) + block.x - 1) / block.x; grid.y = 1; grid.z = 1;
            perioidc_BC_ZY_even << <grid, block >> > (pdf_d);
            cudaErrorCheck(hipDeviceSynchronize()); cudaErrorCheck(hipPeekAtLastError());
            periodic_phi_ZY << <grid, block >> > (4, phi_d);
            cudaErrorCheck(hipDeviceSynchronize()); cudaErrorCheck(hipPeekAtLastError());
        }
        /* ************************** periodic boundary conditions  ***************************************** */

        if (kper == 0 && domain_wall_status_z_min == 0 && domain_wall_status_z_max == 0) { // non - periodic BC along flow direction(z)
            if (inlet_BC == 1) {
                block.x = block_Threads_X;    block.y = block_Threads_Y;    block.z = 1;
                grid.x = (int(nxGlobal) + block.x - 1) / block.x; grid.y = (int(nyGlobal) + block.y - 1) / block.y; grid.z = 1;
                inlet_bounce_back_velocity_BC_before_odd_GPU << <grid, block >> > (walls_d, phi_d, pdf_d, W_in_d);
            }
            else if (inlet_BC == 2) {
                block.x = block_Threads_X;    block.y = block_Threads_Y;    block.z = 1;
                grid.x = (int(nxGlobal) + block.x - 1) / block.x; grid.y = (int(nyGlobal) + block.y - 1) / block.y; grid.z = 1;
                inlet_Zou_He_pressure_BC_before_odd_GPU << <grid, block >> > (rho_in, walls_d, phi_d, pdf_d); // pressure inlet bc
            }
            if (outlet_BC == 1) {
                block.x = block_Threads_X;    block.y = block_Threads_Y;    block.z = 1;
                grid.x = (int(nxGlobal) + block.x - 1) / block.x; grid.y = (int(nyGlobal) + block.y - 1) / block.y; grid.z = 1;
                outlet_convective_BC_before_odd_GPU << <grid, block >> > (walls_d, phi_d, phi_convec_bc_d, pdf_d, f_convec_bc_d, g_convec_bc_d);
            }
            else if (outlet_BC == 2) {
                block.x = block_Threads_X;    block.y = block_Threads_Y;    block.z = 1;
                grid.x = (int(nxGlobal) + block.x - 1) / block.x; grid.y = (int(nyGlobal) + block.y - 1) / block.y; grid.z = 1;
                //grid.x = ceil(double(int(nxGlobal)) / double(block.x));   grid.y = ceil(double(int(nyGlobal)) / double(block.y)); grid.z = 1;
                outlet_Zou_He_pressure_BC_before_odd_GPU << <grid, block >> > (rho_out, walls_d, phi_d, pdf_d); // pressure outlet bc
            }
            cudaErrorCheck(hipDeviceSynchronize()); cudaErrorCheck(hipPeekAtLastError());
        }
        if (porous_plate_cmd != 0) { // place a porous plate
            block.x = block_Threads_X;    block.y = block_Threads_Y;    block.z = 1;
            porous_plate_BC_before_odd << <grid, block >> > (pdf_d);
            cudaErrorCheck(hipDeviceSynchronize()); cudaErrorCheck(hipPeekAtLastError());
        }

    }
    else {
        /* ************************** odd step ***************************************** */
        block.x = block_Threads_X;    block.y = block_Threads_Y;    block.z = block_Threads_Z;
        grid.x = (int(nxGlobal) + block.x - 1) / block.x; grid.y = (int(nyGlobal) + block.y - 1) / block.y; grid.z = (int(nzGlobal) + block.z - 1) / block.z;
        kernel_odd_color_GPU << <grid, block >> > (1, int(nxGlobal), 1, int(nyGlobal), 1, int(nzGlobal), walls_d, pdf_d, phi_d, cn_x_d, cn_y_d, cn_z_d, curv_d, c_norm_d);
        cudaErrorCheck(hipDeviceSynchronize()); cudaErrorCheck(hipPeekAtLastError());
        /* ************************** odd step ***************************************** */
        /* ************************** periodic boundary conditions  ***************************************** */
        if (kper) {
            block.x = block_Threads_X;    block.y = block_Threads_Y;    block.z = 1;
            grid.x = (int(nxGlobal) + block.x - 1) / block.x; grid.y = (int(nyGlobal) + block.y - 1) / block.y; grid.z = 1;
            perioidc_BC_Z_odd << <grid, block >> > (pdf_d);
            cudaErrorCheck(hipDeviceSynchronize()); cudaErrorCheck(hipPeekAtLastError());
            periodic_phi_Z << <grid, block >> > (4, phi_d);
            cudaErrorCheck(hipDeviceSynchronize()); cudaErrorCheck(hipPeekAtLastError());
        }
        if (jper) {
            block.x = block_Threads_X;    block.y = 1;    block.z = block_Threads_Z;
            grid.x = (int(nxGlobal) + block.x - 1) / block.x; grid.y = 1; grid.z = (int(nzGlobal) + block.z - 1) / block.z;
            perioidc_BC_Y_odd << <grid, block >> > (pdf_d);
            cudaErrorCheck(hipDeviceSynchronize()); cudaErrorCheck(hipPeekAtLastError());
            periodic_phi_Y << <grid, block >> > (4, phi_d);
            cudaErrorCheck(hipDeviceSynchronize()); cudaErrorCheck(hipPeekAtLastError());
        }
        if (jper && kper) {
            block.x = block_Threads_X; block.y = 1; block.z = 1;
            grid.x = (int(nxGlobal) + block.x - 1) / block.x; grid.y = 1; grid.z = 1;
            perioidc_BC_ZY_odd << <grid, block >> > (pdf_d);
            cudaErrorCheck(hipDeviceSynchronize()); cudaErrorCheck(hipPeekAtLastError());
            periodic_phi_ZY << <grid, block >> > (4, phi_d);
            cudaErrorCheck(hipDeviceSynchronize()); cudaErrorCheck(hipPeekAtLastError());
        }
        /* ************************** periodic boundary conditions  ***************************************** */

        if (kper == 0 && domain_wall_status_z_min == 0 && domain_wall_status_z_max == 0) { // non - periodic BC along flow direction(z)
            if (inlet_BC == 1) { // velocity inlet bc
                block.x = block_Threads_X;    block.y = block_Threads_Y;    block.z = 1;
                grid.x = (int(nxGlobal) + block.x - 1) / block.x; grid.y = (int(nyGlobal) + block.y - 1) / block.y; grid.z = 1;
                inlet_bounce_back_velocity_BC_after_odd_GPU << <grid, block >> > (walls_d, phi_d, pdf_d, W_in_d);
            }
            else if (inlet_BC == 2) { // pressure inlet bc
                block.x = block_Threads_X;    block.y = block_Threads_Y;    block.z = 1;
                grid.x = (int(nxGlobal) + block.x - 1) / block.x; grid.y = (int(nyGlobal) + block.y - 1) / block.y; grid.z = 1;
                inlet_Zou_He_pressure_BC_after_odd_GPU << <grid, block >> > (rho_in, walls_d, phi_d, pdf_d); // pressure inlet bc
            }
            if (outlet_BC == 1) { // convective outlet bc
                block.x = block_Threads_X;    block.y = block_Threads_Y;    block.z = 1;
                grid.x = (int(nxGlobal) + block.x - 1) / block.x; grid.y = (int(nyGlobal) + block.y - 1) / block.y; grid.z = 1;
                outlet_convective_BC_after_odd_GPU << <grid, block >> >
                    (walls_d, phi_d, phi_convec_bc_d, pdf_d, f_convec_bc_d, g_convec_bc_d);
            }
            else if (outlet_BC == 2) { // pressure outlet bc
                block.x = block_Threads_X;    block.y = block_Threads_Y;    block.z = 1;
                grid.x = (int(nxGlobal) + block.x - 1) / block.x; grid.y = (int(nyGlobal) + block.y - 1) / block.y; grid.z = 1;
                outlet_Zou_He_pressure_BC_after_odd_GPU << <grid, block >> > (rho_out, walls_d, phi_d, pdf_d);   // pressure outlet bc
            }
            cudaErrorCheck(hipDeviceSynchronize()); cudaErrorCheck(hipPeekAtLastError());
        }
        if (porous_plate_cmd != 0) { // place a porous plate
            block.x = block_Threads_X;    block.y = block_Threads_Y;    block.z = 1;
            porous_plate_BC_after_odd << <grid, block >> > (pdf_d);
            cudaErrorCheck(hipDeviceSynchronize()); cudaErrorCheck(hipPeekAtLastError());
        }

    }

    /* Color Gradient */
#pragma region (color gradient)
     /* ~~~~~~~~~~~~~~~~~~~~~~~ extrapolate phi values to solid boundary nodes ~~~~~~~~~~~~~~~~~~ */
    block.x = block_Threads_X;    block.y = block_Threads_Y;    block.z = block_Threads_Z;
    grid.x = (int(nxGlobal) + 6 + block.x - 1) / block.x; grid.y = (int(nyGlobal) + 6 + block.y - 1) / block.y; grid.z = (int(nzGlobal) + 6 + block.z - 1) / block.z;
    extrapolate_phi_toSolid << <grid, block >> > (walls_type_d, phi_d);
    cudaErrorCheck(hipDeviceSynchronize()); cudaErrorCheck(hipPeekAtLastError());
    /* ~~~~~~~~~~~~~~~~~~ calculate normal directions of interfaces from phi gradient ~~~~~~~~~~~~~~~~~~ */
    block.x = block_Threads_X;    block.y = block_Threads_Y;    block.z = block_Threads_Z;
    grid.x = (int(nxGlobal) + 4 + block.x - 1) / block.x; grid.y = (int(nyGlobal) + 4 + block.y - 1) / block.y; grid.z = (int(nzGlobal) + 4 + block.z - 1) / block.z;
    normalDirectionsOfInterfaces << < grid, block >> > (walls_d, phi_d, cn_x_d, cn_y_d, cn_z_d, c_norm_d);
    cudaErrorCheck(hipDeviceSynchronize()); cudaErrorCheck(hipPeekAtLastError());
    /* ~~~~~~~~~~~~~~~ numerically alter the normal directions of interfaces to desired contact angle ~~~~~~~~~~~~~~~~ */
    block.x = block_Threads_X;    block.y = block_Threads_Y;    block.z = block_Threads_Z;
    grid.x = (int(nxGlobal) + 4 + block.x - 1) / block.x; grid.y = (int(nyGlobal) + 4 + block.y - 1) / block.y; grid.z = (int(nzGlobal) + 4 + block.z - 1) / block.z;
    alter_color_gradient_solid_surface_GPU << < grid, block >> > (walls_type_d, cn_x_d, cn_y_d, cn_z_d, c_norm_d, s_nx_d, s_ny_d, s_nz_d);
    cudaErrorCheck(hipDeviceSynchronize()); cudaErrorCheck(hipPeekAtLastError());
    /* ~~~~~~~~~~~~~~ extrapolate normal direction info to solid boundary nodes, to minimize unbalanced forces ~~~~~~~~~~~~~~ */
    block.x = block_Threads_X;    block.y = block_Threads_Y;    block.z = block_Threads_Z;
    grid.x = (int(nxGlobal) + 2 + block.x - 1) / block.x; grid.y = (int(nyGlobal) + 2 + block.y - 1) / block.y; grid.z = (int(nzGlobal) + 2 + block.z - 1) / block.z;
    extrapolateNormalToSolid << <grid, block >> > (walls_type_d, cn_x_d, cn_y_d, cn_z_d, phi_d);
    cudaErrorCheck(hipDeviceSynchronize()); cudaErrorCheck(hipPeekAtLastError());
    /* ~~~~~~~~~~~~~~~~~~ calculate CSF forces based on interace curvature  ~~~~~~~~~~~~~~~~~~ */
    //block.x = block_Threads_X;    block.y = block_Threads_Y;    block.z = block_Threads_Z;
    block.x = block_Threads_X;    block.y = block_Threads_Y;    block.z = block_Threads_Z;
    grid.x = (int(nxGlobal) + block.x - 1) / block.x; grid.y = (int(nyGlobal) + block.y - 1) / block.y; grid.z = (int(nzGlobal) + block.z - 1) / block.z;
    CSF_Forces << <grid, block >> > (cn_x_d, cn_y_d, cn_z_d, curv_d);
    cudaErrorCheck(hipDeviceSynchronize()); cudaErrorCheck(hipPeekAtLastError());
#pragma endregion (color gradient)


    /* send results back to the host side */
    if (ntime % ntime_monitor == 0 || ntime % ntime_animation == 0 || ntime % ntime_visual == 0 || ntime % ntime_clock_sum == 0) {
        cout << "Copying results back to the host side .... ";
        cudaErrorCheck(hipMemcpy(u, u_d, mem_size_s1_TP, hipMemcpyDeviceToHost));
        cudaErrorCheck(hipMemcpy(v, v_d, mem_size_s1_TP, hipMemcpyDeviceToHost));
        cudaErrorCheck(hipMemcpy(w, w_d, mem_size_s1_TP, hipMemcpyDeviceToHost));
        cudaErrorCheck(hipMemcpy(rho, rho_d, mem_size_s1_TP, hipMemcpyDeviceToHost));
        cudaErrorCheck(hipMemcpy(phi, phi_d, mem_size_s4_TP, hipMemcpyDeviceToHost));
        cudaErrorCheck(hipMemcpy(curv, curv_d, mem_size_s1_TP, hipMemcpyDeviceToHost));
        cudaErrorCheck(hipMemcpy(c_norm, c_norm_d, mem_size_s2_TP, hipMemcpyDeviceToHost));
        cudaErrorCheck(hipMemcpy(cn_x, cn_x_d, mem_size_s2_TP, hipMemcpyDeviceToHost));
        cudaErrorCheck(hipMemcpy(cn_y, cn_y_d, mem_size_s2_TP, hipMemcpyDeviceToHost));
        cudaErrorCheck(hipMemcpy(cn_z, cn_z_d, mem_size_s2_TP, hipMemcpyDeviceToHost));
        
        cudaErrorCheck(hipMemcpy(pdf, pdf_d, mem_size_f1_TP, hipMemcpyDeviceToHost));

        cudaErrorCheck(hipDeviceSynchronize()); cudaErrorCheck(hipPeekAtLastError());
        cout << " Complete " << endl;
    }

}





